#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>

#define SIZE 32

using namespace std;

__device__ unsigned int device_data[SIZE];

__global__ void	ParalelRadixSort()
{
	//sprememba
	__shared__ volatile unsigned int shared_data[SIZE * 2];

	shared_data[threadIdx.x] = device_data[threadIdx.x];
		
	unsigned int bitmask = 1 << 0;
	unsigned int offset = 0;
	// -1, -2, -4, -8, -16, -32, -64, -128, -256,...
	unsigned int thrmask = 0xFFFFFFFFU << threadIdx.x;
	unsigned int mypos;

	for (int i = 0; i <= 10; i++)
	{
		unsigned int mydata = shared_data[((SIZE - 1) - threadIdx.x) + offset];
		unsigned int mybit = mydata&bitmask;
		// Get population of ones and zeroes
		unsigned int ones = __ballot(mybit);
		unsigned int zeroes = ~ones;
		// Switch ping-pong buffers
		offset ^= SIZE;

		// Do zeroes, then ones
		if (!mybit)
		{
			mypos = __popc(zeroes&thrmask);
		}
		else {      // Threads with a one bit
					// Get my position in ping-pong buffer
			mypos = __popc(zeroes) + __popc(ones&thrmask);
		}

		// Move to buffer  (or use shfl for cc 3.0)
		shared_data[mypos - 1 + offset] = mydata;
		// Repeat for next bit
		bitmask <<= 1;

		device_data[threadIdx.x] = shared_data[threadIdx.x + offset];
	}
		

}
void WriteFile(vector<int> vector) {
	ofstream myfile;
	myfile.open("out.txt");
	for each (int var in vector)
	{
		myfile << var << " ";
	}
	myfile.close();
}
vector<int> ReadFile(string filepath) {
	vector<int> vector;
	ifstream inputFile(filepath);
	double value;

	// read the elements in the file into a vector  
	while (inputFile >> value) {
		vector.push_back(value);
	}
	for each (int var in vector)
	{
		cout << var << " ";
	}
	inputFile.close();
	return vector;
}

int main(int argc, char * argv[])
{
	cout << argv[1] << endl;

	std::ifstream fin(argv[1]);
	

	vector<int> vector;
	

	unsigned int hdata[SIZE];
	float totalTime = 0;
	float milliseconds = 0;

	//Time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (!fin) {
		cerr << "Datoteka ne obstaja " << argv[1] << endl;
	}
	else {
		cout << "Datoteka obstaja " << argv[1] << endl;
		vector = ReadFile(argv[1]);

		WriteFile(vector);
	}

	int size = vector.size();
	//vector.resize(size);

	std::copy(vector.begin(), vector.end(), hdata);
	//int *a = vector.data();

	// Copy data from host to device
	hipMemcpyToSymbol(HIP_SYMBOL(device_data), hdata, SIZE * sizeof(unsigned int));

	// Execution time measurement, that point starts the clock
	hipEventRecord(start);
	ParalelRadixSort << < 1, SIZE >> >();
		
	// Execution time measurement, that point stops the clock
	hipEventRecord(stop);

	// Make kernel function synchronous
	hipDeviceSynchronize();

	hipEventElapsedTime(&milliseconds, start, stop);
	totalTime += milliseconds;
	//milliseconds = 0;

	// Copy data from device to host
	hipMemcpyFromSymbol(hdata, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));

	std::vector<int> v(hdata, hdata + sizeof hdata / sizeof hdata[0]);
	v.resize(size);

	/*printf("TIME %4.2fs", milliseconds);
	printf("\n");
	printf("Effective Bandwidth (GB/s): %fn", SIZE * 4 * 3 / milliseconds / 1e6);
	printf("\n");*/
	cout << endl;
	cout << "Host Data" << endl;
	for each (int var in v)
	{
		cout << var << " ";
	}
	/*cout << "Device Data" << endl;
	for each (int var in device_data)
	{
		cout << var << " ";
	}
	*/
	cout << endl;
	cout << "TIME %4.2fs " << milliseconds << endl;

    return 0;
}



