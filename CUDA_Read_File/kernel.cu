#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""

#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>

#define SIZE 32 //512
#define BIT 32

using namespace std;

__device__ unsigned int device_data[SIZE];
__device__ int ddata_s[SIZE];

//Test v gpu in nazaj
__global__ void gpucopy(int* src, int* dst)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;;
	dst[i] = src[i];
}

__global__ void radixSort(int* sort_tmp, const int num_lists, const int num_element, int* sort_tmp_0, int* sort_tmp_1) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	//printf("\n%d num list: %d" ,tid, num_lists );
	//printf("\n num element: %d" , num_element);
	if (sort_tmp == NULL) {
		printf("\nsort_tmp is null");
	}
	
	
	for (int bit = 0; bit < BIT; bit++) {
		int base_cnt_0 = 0, base_cnt_1 = 0;
		//printf("\ntest1 + %d",tid );
		for (int i = 0; i < num_element; i+=num_lists) {
			int elem = sort_tmp[i + tid];
			const int bit_mask = (1 << bit);
			//printf("\ntest1 + %d", elem);
			if ((elem & bit_mask) > 0) {
				sort_tmp_1[base_cnt_1 + tid] = elem;
				base_cnt_1 += num_lists;
			}
			else {
				sort_tmp_0[base_cnt_0 + tid] = elem;
				base_cnt_0+=num_lists;
			}
		}
		//copy data back to source - first the zero list
		for (int i = 0; i < base_cnt_0; i += num_lists) {
			sort_tmp[i + tid] = sort_tmp_0[i + tid];
		}
		//copy data back to source - then the one list
		for (int i = 0; i < base_cnt_1; i += num_lists) {
			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
		}
	}
	
	__syncthreads();
	/*for (int i = 0; i < num_element; i++) {
		printf("\n%d", sort_tmp[i]);
	}*/
}


void WriteFile(vector<double> vector) {
	ofstream myfile;
	myfile.open("out.txt");
	for each (double var in vector)
	{
		myfile << var << " ";
	}
	myfile.close();
}
vector<int> ReadFile(string filepath) {
	vector<int> vector;
	ifstream inputFile(filepath);
	double value;

	// read the elements in the file into a vector  
	while (inputFile >> value) {
		vector.push_back(value * 100);
	}
	for each (int var in vector)
	{
		cout << var << " ";
	}
	inputFile.close();
	return vector;
}

int main(int argc, char * argv[])
{
	cout << argv[1] << endl;

	std::ifstream fin(argv[1]);
	
	std::vector<int> host_double(7);
	std::vector<int> host_double2(7);

	float totalTime = 0;
	float milliseconds = 0;

	//Time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (!fin) {
		cerr << "Datoteka ne obstaja " << argv[1] << endl;
	}
	else {
		cout << "Datoteka obstaja " << argv[1] << endl;
		host_double = ReadFile(argv[1]);

		//WriteFile(host_double);
	}
	int size = host_double.size();
	
	int* device = NULL;
	int* tmp_0 = NULL;
	int* tmp_1 = NULL;
	
	hipMalloc((void**)&device, size * sizeof(int));
	hipMalloc((void**)&tmp_0, size * sizeof(int));
	hipMalloc((void**)&tmp_1, size * sizeof(int));

	int* a = host_double.data();
	

	//num_list 32 

	hipMemcpy(device,&*host_double.data(), size * sizeof(int), hipMemcpyHostToDevice);
	
	radixSort << < 1,SIZE >> >(device, SIZE, size, tmp_0, tmp_1);
	//ParalelRadixSort << <(size + size - 1) / size, size >> >();
	//gpucopy << <1,SIZE>> >(device, device2);
	
	hipDeviceSynchronize();

	hipMemcpy(&*host_double.data(), device, size * sizeof(int), hipMemcpyDeviceToHost);

	cout << endl << "host data" << endl;
	for each (int var in host_double)
	{
		double d = (double)var / 100;
		cout << d << " " << endl;
	}

	hipFree(device);
	hipFree(tmp_0);
	hipFree(tmp_1);

	//std::vector<int> v(10);

	//unsigned int hdata[SIZE];


	//	int size = host_double.size();
	//	//vector.resize(size);

	//	std::copy(host_double.begin(), host_double.end(), hdata);
	//	//int *a = vector.data();

	//	// Copy data from host to device
	//	hipMemcpyToSymbol(HIP_SYMBOL(device_data), hdata, SIZE * sizeof(unsigned int));

	//	// Execution time measurement, that point starts the clock
	//	hipEventRecord(start);
	//	ParalelRadixSort << <1, SIZE>> >();

	//	// Execution time measurement, that point stops the clock
	//	hipEventRecord(stop);

	//	// Make kernel function synchronous
	//	hipDeviceSynchronize();

	//	hipEventElapsedTime(&milliseconds, start, stop);
	//	totalTime += milliseconds;
	//	//milliseconds = 0;

	//	// Copy data from device to host
	//	hipMemcpyFromSymbol(hdata, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));

	//	/*std::vector<int> v(hdata, hdata + sizeof hdata / sizeof hdata[0]);
	//	v.resize(size);*/
	//	milliseconds = 0;
	//

	///*printf("TIME %4.2fs", milliseconds);
	//printf("\n");
	//printf("Effective Bandwidth (GB/s): %fn", SIZE * 4 * 3 / milliseconds / 1e6);
	//printf("\n");*/
	//cout << endl;
	//cout << "Host Data" << endl;
	///*for each (int var in v)
	//{
	//	double d = (double)var / 100;
	//	cout << d << " " << endl;
	//}*/
	//for (int i = 0; i < SIZE; i++) {
	//	double d = (double)hdata[i] / 100;
	//	cout << d << endl;
	//}
	//
	//cout << endl;
	//cout << "TIME %4.2fs " << totalTime << endl;

    return 0;
}

