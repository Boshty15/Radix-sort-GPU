#include "hip/hip_runtime.h"
//
//
//
//#pragma once
//#ifdef __INTELLISENSE__
//void __syncthreads();
//#endif


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>
#include <bitset>
#include <time.h>
#include <ctime>
#include <chrono>
#include <math.h> 

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <algorithm>
#include <cstdlib>
#include <Windows.h>

using namespace std;
using namespace std::chrono;

int getMax(unsigned int* arr, int n)
{
	int mx = arr[0];
	for (int i = 1; i < n; i++)
		if (arr[i] > mx)
			mx = arr[i];
	return mx;
}

void countSort(unsigned int* arr, int n, int exp)
{
	// output array
	vector<unsigned int> ve(n);
	unsigned int* output = ve.data();
	int i, count[10] = { 0 };

	// Store count of occurrences in count[]
	for (i = 0; i < n; i++)
		count[(arr[i] / exp) % 10]++;

	// Change count[i] so that count[i] now contains actual
	//  position of this digit in output[]
	for (i = 1; i < 10; i++)
		count[i] += count[i - 1];

	// Build the output array
	for (i = n - 1; i >= 0; i--)
	{
		output[count[(arr[i] / exp) % 10] - 1] = arr[i];
		count[(arr[i] / exp) % 10]--;
	}

	// Copy the output array to arr[], so that arr[] now
	// contains sorted numbers according to current digit
	for (i = 0; i < n; i++)
		arr[i] = output[i];
}

// Radix Sort
void serialRadixSort(unsigned int* arr, int n)
{
	// Find the maximum number to know number of digits
	int m = getMax(arr, n);

	// Do counting sort for every digit. Note that instead
	// of passing digit number, exp is passed. exp is 10^i
	// where i is current digit number
	for (int exp = 1; m / exp > 0; exp *= 10)
		countSort(arr, n, exp);
}


int main(int argc, char * argv[]) {

	//host
	//thrust::sort(thrust::host, A, A + N);

	unsigned long long int N;
	cout << "Izberi �tevilo elementov:" << endl << endl;
	cout << "(1) 1M elementov:  " << endl;
	cout << "(2) 5M elementov:  " << endl;
	cout << "(3) 10M elementov:  " << endl;
	cout << "(4) 25M elementov:  " << endl;
	cout << "(5) 50M elementov:  " << endl;
	cout << "(6) 75M elementov:  " << endl;
	cout << "(7) 100M elementov:  " << endl;
	cin >> N;

	switch (N)
	{
	case 1:
		N = 1000000;
		break;
	case 2:
		N = 5000000;
		break;
	case 3:
		N = 10000000;
		break;
	case 4:
		N = 25000000;
		break;
	case 5:
		N = 50000000;
		break;
	case 6:
		N = 75000000;
		break;
	case 7:
		N = 100000000;
		break;
	default:
		break;
	}
	cout << "Izbral si: " << N << " elementov" << endl;
	
	//////thrust parralel
	cout << "Parralel Radix sort Thrust" << endl;

	thrust::host_vector<unsigned int> host_int_Parralel(N);
	std::generate(host_int_Parralel.begin(), host_int_Parralel.end(), rand);

	cout << "Parralel size: " << host_int_Parralel.size() << endl;

	thrust::device_vector<unsigned int> d_vec = host_int_Parralel;

	high_resolution_clock::time_point start = high_resolution_clock::now();
	// sort data on the device (846M keys per second on GeForce GTX 480)
	thrust::sort(d_vec.begin(), d_vec.end());
	high_resolution_clock::time_point stop = high_resolution_clock::now();

	// transfer data back to host
	thrust::copy(d_vec.begin(), d_vec.end(), host_int_Parralel.begin());

	auto durationParralel = duration_cast <milliseconds> (stop - start).count();
	cout << "Time: " << durationParralel << " milliseconds" << endl;



	cout << "Serial Radix sort Thrust" << endl;

	thrust::host_vector<unsigned int> host_int_Serial(N);
	std::generate(host_int_Serial.begin(), host_int_Serial.end(), rand);

	cout << "Serial size: " << host_int_Serial.size() << endl;

	//thrust::device_vector<unsigned long int> d_vec_S;
	unsigned int* da = host_int_Serial.data();
	high_resolution_clock::time_point startS = high_resolution_clock::now();
	// sort data on the device (846M keys per second on GeForce GTX 480)
	serialRadixSort(da, N);
	high_resolution_clock::time_point stopS = high_resolution_clock::now();
	
	auto durationS = duration_cast <milliseconds> (stopS - startS).count();
	cout << "Time: " << durationS << " milliseconds" << endl;
	
	 //sort data on the device (846M keys per second on GeForce GTX 480)
	//thrust::sort(thrust::host, d_vec_S.begin(), d_vec_S.end());
	
	
	 //transfer data back to host
	//thrust::copy(d_vec_S.begin(), d_vec_S.end(), host_int_Serial.begin());
//	auto durationSerial = duration_cast <milliseconds> (stopS - startS).count();
	/*for (int i = 0; i < N; i++) {
		if (da[i] > da[i + 1]) {
			printf("sort error at, hdata[%d] = %d, hdata[%d] = %d\n", i, da[i], i + 1, da[i + 1]);
			return 1;
		}
	}*/



	

	 
	

}





//
//#define BIT 32
//#define SIZE (33 * 1024)
//
///*Serial radix sort*/



//__device__ void radixSortParralel(unsigned int* sort_tmp, const int num_lists, const int num_element, const int tid, unsigned int* sort_tmp_1) {
//	//if (sort_tmp == NULL) {
//	//	printf("\nsort_tmp is null");
//	//}
//
//	for (int bit = 0; bit < BIT; bit++) {
//		const int bit_mask = (1 << bit);
//		int base_cnt_0 = 0, base_cnt_1 = 0;
//		//printf("\ntest1 + %d",tid );
//		for (int i = 0; i < num_element; i += num_lists) {
//			int elem = sort_tmp[i + tid];
//
//			//printf("\ntest %d + %d bit %d",tid, elem, bit_mask);
//			if ((elem & bit_mask) > 0) {
//				sort_tmp_1[base_cnt_1 + tid] = elem;
//				base_cnt_1 += num_lists;
//				//printf("\n element 1 %d = %d",i, elem);
//			}
//			else {
//				sort_tmp[base_cnt_0 + tid] = elem;
//				base_cnt_0 += num_lists;
//				//printf("\n element 0 %d = %d",i, elem);
//			}
//		}
//		//copy data back to source - first the zero list
//		for (int i = 0; i < base_cnt_1; i += num_lists) {
//			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
//			//printf("\nsort tmp: %d", sort_tmp_0[i + tid]);
//		}
//	}
//	__syncthreads();
//
//	//for (int i = 0; i < num_element; i++) {
//	//	printf("\n%d", sort_tmp[i]);
//	//}
//}
//int find_min(unsigned int * const src_array,
//	int * const list_indexes,
//	const int num_lists,
//	const int num_elements_per_list)
//{
//	int min_val = 0xFFFFFFFF;
//	int min_idx = 0;
//	// Iterate over each of the lists
//	for (int i = 0; i < num_lists; i++)
//	{
//		// If the current list has already been emptied
//		// then ignore it
//		if (list_indexes[i] < num_elements_per_list)
//		{
//			const int src_idx = i + (list_indexes[i] * num_lists);
//			const int data = src_array[src_idx];
//			if (data <= min_val)
//			{
//				min_val = data;
//				min_idx = i;
//			}
//		}
//	}
//	list_indexes[min_idx]++;
//	return min_val;
//}
//__device__ void copy_data_to_shared(unsigned int * data,
//	unsigned int * sort_tmp,
//	const int num_lists,
//	const int num_elements,
//	const int tid)
//{
//	// Copy data into temp store
//	for (int i = 0; i < num_elements; i += num_lists)
//	{
//		sort_tmp[i + tid] = data[i + tid];
//	}
//	__syncthreads();
//}
//__device__ void merge_array6(unsigned int * src_array,
//	unsigned int * dest_array,
//	const int num_lists,
//	const int num_elements,
//	const int tid)
//{
//	const int num_elements_per_list = (num_elements / num_lists);
//	
//	__shared__ int list_indexes[32];
//
//	list_indexes[tid] = 0;
//	// Wait for list_indexes[tid] to be cleared
//	__syncthreads();
//	// Iterate over all elements
//	for (int i = 0; i < num_elements; i++)
//	{
//		// Create a value shared with the other threads
//		__shared__ int min_val;
//		__shared__ int min_tid;
//		// Use a temp register for work purposes
//		int data;
//		// If the current list has not already been
//		// emptied then read from it, else ignore it
//		if (list_indexes[tid] < num_elements_per_list)
//		{
//			// Work out from the list_index, the index into
//			// the linear array
//			const int src_idx = tid + (list_indexes[tid] * num_lists);
//			// Read the data from the list for the given
//			// thread
//			data = src_array[src_idx];
//		}
//		else
//		{
//			data = 0xFFFFFFFF;
//		}
//		// Have thread zero clear the min values
//		if (tid == 0)
//		{
//			// Write a very large value so the first
//			// thread thread wins the min
//			min_val = 0xFFFFFFFF;
//			min_tid = 0xFFFFFFFF;
//		}
//		// Wait for all threads
//		__syncthreads();
//		// Have every thread try to store it�s value into
//		// min_val. Only the thread with the lowest value
//		// will win
//
//		atomicMin(&min_val, data);
//
//		// Make sure all threads have taken their turn.
//		__syncthreads();
//		// If this thread was the one with the minimum
//		if (min_val == data)
//		{
//			// Check for equal values
//			// Lowest tid wins and does the write
//
//			atomicMin(&min_tid, tid);
//		}
//		// Make sure all threads have taken their turn.
//		__syncthreads();
//		// If this thread has the lowest tid
//		if (tid == min_tid)
//		{
//			// Incremene the list pointer for this thread
//			list_indexes[tid]++;
//			// Store the winning value
//			dest_array[i] = data;
//		}
//	}
//}
//
//__global__ void gpu_sort_array_array(unsigned int * data, const int num_lists, const int num_elements)
//{
//	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
//
//	__shared__ unsigned int sort_tmp[64];
//	__shared__ unsigned int sort_tmp_1[64];
//
//	copy_data_to_shared(data, sort_tmp, num_lists,
//		num_elements, tid);
//	radixSortParralel(sort_tmp, num_lists, num_elements,
//		tid, sort_tmp_1);
//	merge_array6(sort_tmp, data, num_lists,
//		num_elements, tid);
//
//}
//
//void print(unsigned int* arr, int n)
//{
//	for (int i = 0; i < n; i++)
//		cout << (double)arr[i] / 100 << endl;
//}
//
//void menu() {
//	cout << "Izberi: " << endl;
//	cout << " 0 Exit " << endl;
//	cout << " 1 Serial sort" << endl;
//	cout << " 2 Parrallel sort" << endl;
//	cout << " 3 Thrust Parrallel sort" << endl;
//}
//
////void WriteFile(vector<double> vector) {
////	ofstream myfile;
////	myfile.open("out.txt");
////	for each (double var in vector)
////	{
////		myfile << var << " ";
////	}
////	myfile.close();
////}
//
//vector<unsigned int> ReadFile(string filepath) {
//	vector<unsigned int> vector;
//	ifstream inputFile(filepath);
//	double value;
//
//	//read the elements in the file into a vector  
//	while (inputFile >> value) {
//		vector.push_back(value * 100);
//	}
//	inputFile.close();
//	return vector;
//}
//
//int main(int argc, char * argv[]) {
//
//	int blockSize = 32;
//	int numBlocks = (SIZE + blockSize - 1) / blockSize;
//
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	float totalTime = 0;
//
//	std::ifstream fin(argv[1]);
//	//std::vector<unsigned int> host_double(1);
//
//	unsigned int host_data[SIZE];
//
//	// generate 32M random numbers serially
//	
//
//	if (!fin) {
//		cerr << "Datoteka ne obstaja " << argv[1] << endl;
//	}
//	else {
//		cout << "Datoteka obstaja " << argv[1] << endl;
//		//host_double = ReadFile(argv[1]);
//
//		//WriteFile(host_double);
//	}
//	
//
//	while (true) {
//		thrust::host_vector<unsigned int> host_double(1 << 20);
//		std::generate(host_double.begin(), host_double.end(), rand);
//		int size = host_double.size();
//
//		int izbiraAlg;
//		menu();
//		cin >> izbiraAlg;
//		if (izbiraAlg == 0) {
//			return 0;
//		}
//		else if (izbiraAlg == 1) {
//
//			//Serial radix sort
//
//			unsigned int* da = host_double.data();
//			int n = sizeof(host_double) / sizeof(host_double[0]);
//			//auto t1 = high_resolution_clock::now();
//			high_resolution_clock::time_point start = high_resolution_clock::now();
//			serialRadixSort(da, size);
//			high_resolution_clock::time_point stop = high_resolution_clock::now();
//			auto duration = duration_cast <milliseconds> (stop - start).count();
//			cout << endl << "Serial radix sort " << endl;
//			//print(da, size);
//			for (int i = 0; i < size - 1; i++) {
//				if (host_double[i] > host_double[i + 1]) {
//					printf("sort error at, hdata[%d] = %d, hdata[%d] = %d\n", i, host_double[i], i + 1, host_double[i + 1]);
//					return 1;
//				}
//			}
//			cout << "Total time: " << duration << "ms" << endl;
//			cout << "Success" << endl;
//
//		}
//		else if (izbiraAlg == 2) {
//
//			//Parrallel radix sort
//			cout << endl;
//			cout << "Parrallel sort " << endl;
//			std::copy(host_double.begin(), host_double.end(), host_data);
//			/*cout << "Not sorted!" << endl;
//			print(host_data, size);*/
//
//			unsigned int* ddata;
//			unsigned int* ddata_tmp;
//			//
//
//			//for (int lcount = 0; lcount < LOOPS; lcount++) {
//			//	unsigned int range = 1U << UPPER_BIT;
//			//	//for (int i = 0; i < SIZE; i++) host_data[i] = rand() % range;
//				//hipMemcpyToSymbol(HIP_SYMBOL(device_data), host_data, SIZE * sizeof(unsigned int));
//			hipMalloc((void**)&ddata_tmp, SIZE * sizeof(unsigned int));
//			hipMalloc((void**)&ddata, SIZE * sizeof(unsigned int));
//			hipMemcpy(ddata, host_data, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
//
//			high_resolution_clock::time_point start = high_resolution_clock::now();
//			//radixSortParralel << <1, 10 >> >(ddata, ddata_tmp);
//			//radixSort << <(numBlocks, blockSize) >> > (ddata, ddata_tmp);
//			gpu_sort_array_array << < 1, 2 >> > (ddata, 32, 64);
//			high_resolution_clock::time_point stop = high_resolution_clock::now();
//			host_data[SIZE];
//
//			hipMemcpy(host_data, ddata, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
//
//			//hipMemcpyFromSymbol(host_data, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));
//			auto durationParralel = duration_cast <milliseconds> (stop - start).count();
//			//	/*for (int i = 0; i < SIZE - 1; i++) {
//			//	if (host_data[i] > host_data[i + 1]) {
//			//	printf("sort error at loop %d, hdata[%d] = %d, hdata[%d] = %d\n", lcount, i, host_data[i], i + 1, host_data[i + 1]);
//			//	return 1;
//			//	}
//			//	}*/
//			//	cout << " Sorted data: " << endl;
//			//	for (int i = 0; i < size; i++) {
//			//		cout << i << "     " <<(double)host_data[i] / 100 << endl;
//			//	}
//			//	cout << " Time parralel sort: " << durationParralel << endl;
//			//}
//			cout << endl << "Sorted" << endl;
//			print(host_data, size);
//
//		}
//		else {
//			//thrust
//
//			cout << "Parralel Radix sort Thrust" << endl;
//
//			// transfer data to the device
//			thrust::device_vector<int> d_vec = host_double;
//
//			high_resolution_clock::time_point start = high_resolution_clock::now();
//			// sort data on the device (846M keys per second on GeForce GTX 480)
//			thrust::sort(d_vec.begin(), d_vec.end());
//			high_resolution_clock::time_point stop = high_resolution_clock::now();
//
//			// transfer data back to host
//			thrust::copy(d_vec.begin(), d_vec.end(), host_double.begin());
//
//			auto durationParralel = duration_cast <milliseconds> (stop - start).count();
//			cout << "Time: " << durationParralel << "ms" << endl;
//			/*for each (unsigned int var in host_double)
//			{
//				double tmp = (double)var / 100;
//				cout << tmp << endl;
//			}*/
//		}
//	}
//
//
//	return 0;
//}


