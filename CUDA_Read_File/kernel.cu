#include "hip/hip_runtime.h"
//
//
//
//#pragma once
//#ifdef __INTELLISENSE__
//void __syncthreads();
//#endif


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>
#include <bitset>
#include <time.h>
#include <ctime>
#include <chrono>
#include <math.h> 

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <algorithm>
#include <cstdlib>
#include <Windows.h>

using namespace std;
using namespace std::chrono;

size_t getMax2(vector<string> arr, int n) {
	size_t max = arr[0].size();
	for (int i = 1; i < n; i++) {
		if (arr[i].size()>max)
			max = arr[i].size();
	}
	return max;
}

void countSort2(vector<string> a, int size, size_t k) {
	string *b = NULL; int *c = NULL;
	b = new string[size];
	c = new int[257];



	for (int i = 0; i <257; i++) {
		c[i] = 0;
		//cout << c[i] << "\n";
	}
	for (int j = 0; j <size; j++) {
		c[k < a[j].size() ? (int)(unsigned char)a[j][k] + 1 : 0]++;            //a[j] is a string
																			   //cout << c[a[j]] << endl;
	}

	for (int f = 1; f <257; f++) {
		c[f] += c[f - 1];
	}

	for (int r = size - 1; r >= 0; r--) {
		b[c[k < a[r].size() ? (int)(unsigned char)a[r][k] + 1 : 0] - 1] = a[r];
		c[k < a[r].size() ? (int)(unsigned char)a[r][k] + 1 : 0]--;
	}

	for (int l = 0; l < size; l++) {
		a[l] = b[l];
	}

	// avold memory leak
	delete[] b;
	delete[] c;
}


void radixSort2(vector<string> b, int r) {
	size_t max = getMax2(b, r);
	for (size_t digit = max; digit > 0; digit--) { // size_t is unsigned, so avoid using digit >= 0, which is always true
		countSort2(b, r, digit - 1);
	}

}



template <class T>
T getMax(T* arr, int n)
{
	T mx = arr[0];
	for (int i = 1; i < n; i++)
		if (arr[i] > mx)
			mx = arr[i];
	return mx;
}

template <class T>
void countSort(T* arr, int n, int exp)
{
	// output array
	vector<T> ve(n);
	T* output = ve.data();
	int i, count[10] = { 0 };

	// Store count of occurrences in count[]
	for (i = 0; i < n; i++)
		count[(arr[i] / exp) % 10]++;

	// Change count[i] so that count[i] now contains actual
	//  position of this digit in output[]
	for (i = 1; i < 10; i++)
		count[i] += count[i - 1];

	// Build the output array
	for (i = n - 1; i >= 0; i--)
	{
		output[count[(arr[i] / exp) % 10] - 1] = arr[i];
		count[(arr[i] / exp) % 10]--;
	}

	// Copy the output array to arr[], so that arr[] now
	// contains sorted numbers according to current digit
	for (i = 0; i < n; i++)
		arr[i] = output[i];
}

// Radix Sort
template <class T>
void serialRadixSort(T* arr, int n)
{
	// Find the maximum number to know number of digits
	T m = getMax(arr, n);

	// Do counting sort for every digit. Note that instead
	// of passing digit number, exp is passed. exp is 10^i
	// where i is current digit number
	for (int exp = 1; m / exp > 0; exp *= 10)
		countSort(arr, n, exp);
}


int main(int argc, char * argv[]) {

	//host
	//thrust::sort(thrust::host, A, A + N);

	unsigned long long int N;
	cout << "Izberi �tevilo elementov:" << endl << endl;
	cout << "(1) 1M elementov:  " << endl;
	cout << "(2) 5M elementov:  " << endl;
	cout << "(3) 10M elementov:  " << endl;
	cout << "(4) 25M elementov:  " << endl;
	cout << "(5) 50M elementov:  " << endl;
	cout << "(6) 75M elementov:  " << endl;
	cout << "(7) 100M elementov:  " << endl;
	cin >> N;

	switch (N)
	{
	case 1:
		N = 1000000;
		break;
	case 2:
		N = 5000000;
		break;
	case 3:
		N = 10000000;
		break;
	case 4:
		N = 25000000;
		break;
	case 5:
		N = 50000000;
		break;
	case 6:
		N = 75000000;
		break;
	case 7:
		N = 100000000;
		break;
	default:
		break;
	}
	cout << "Izbral si: " << N << " elementov" << endl;


	cout << "(1) Integer" << endl;
	cout << "(2) Char" << endl;
	cout << "(3) Long" << endl;
	cout << "(4) Float" << endl;
	cout << "(5) Double" << endl;
	int tip;
	cin >> tip;

	switch (tip)
	{
	case 1: 
	{
		cout << endl;
		cout << endl;
		cout << "Int" << endl << endl;

		

		//cout << "Parralel: HOST Stable Radix sort" << endl;

		//thrust::host_vector<unsigned int> host_int_Parralel_Host(N);
		//std::generate(host_int_Parralel_Host.begin(), host_int_Parralel_Host.end(), rand);

		//cout << "Parralel size: " << host_int_Parralel_Host.size() << endl;

		//thrust::device_vector<unsigned int> d_vec_Host = host_int_Parralel_Host;

		//high_resolution_clock::time_point start_Host = high_resolution_clock::now();
		//// sort data on the device (846M keys per second on GeForce GTX 480)
		//thrust::stable_sort(thrust::host,d_vec_Host.begin(), d_vec_Host.end());
		//high_resolution_clock::time_point stop_Host = high_resolution_clock::now();

		//// transfer data back to host
		//thrust::copy(d_vec_Host.begin(), d_vec_Host.end(), host_int_Parralel_Host.begin());

		//auto durationParralel_Host = duration_cast <milliseconds> (stop_Host - start_Host).count();
		//cout << "Time: " << durationParralel_Host << " milliseconds" << endl;

		//cout << endl;
		//cout << endl;


		cout << "Parralel:  Stable Radix sort" << endl;

		thrust::host_vector<unsigned int> host_int_Parralel_S(N);
		std::generate(host_int_Parralel_S.begin(), host_int_Parralel_S.end(), rand);

		cout << "Parralel size: " << host_int_Parralel_S.size() << endl;

		thrust::device_vector<unsigned int> d_vec_S = host_int_Parralel_S;

		high_resolution_clock::time_point start_S = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::stable_sort(d_vec_S.begin(), d_vec_S.end());
		high_resolution_clock::time_point stop_S = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec_S.begin(), d_vec_S.end(), host_int_Parralel_S.begin());

		auto durationParralel_S = duration_cast <milliseconds> (stop_S - start_S).count();
		cout << "Time: " << durationParralel_S << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		//////thrust parralel
		cout << "Parralel: Radix sort" << endl;

		thrust::host_vector<unsigned int> host_int_Parralel(N);
		std::generate(host_int_Parralel.begin(), host_int_Parralel.end(), rand);

		cout << "Parralel size: " << host_int_Parralel.size() << endl;

		thrust::device_vector<unsigned int> d_vec = host_int_Parralel;

		high_resolution_clock::time_point start = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::sort(d_vec.begin(), d_vec.end());
		high_resolution_clock::time_point stop = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec.begin(), d_vec.end(), host_int_Parralel.begin());

		auto durationParralel = duration_cast <milliseconds> (stop - start).count();
		cout << "Time: " << durationParralel << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		cout << "Serial Radix sort" << endl;

		thrust::host_vector<unsigned int> host_int_Serial(N);
		std::generate(host_int_Serial.begin(), host_int_Serial.end(), rand);

		cout << "Serial size: " << host_int_Serial.size() << endl;

		//thrust::device_vector<unsigned long int> d_vec_S;
		unsigned int* da = host_int_Serial.data();
		high_resolution_clock::time_point startS = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		serialRadixSort<unsigned int>(da, N);
		high_resolution_clock::time_point stopS = high_resolution_clock::now();

		auto durationS = duration_cast <milliseconds> (stopS - startS).count();
		cout << "Time: " << durationS << " milliseconds" << endl;

		cout << endl;
		cout << endl; 
		break;
	}
	case 2:
	{
		cout << endl;
		cout << endl;
		cout << "Char" << endl << endl;

		cout << "Parralel:  Stable Radix sort" << endl;

		thrust::host_vector<char> host_int_Parralel_S(N);
		std::generate(host_int_Parralel_S.begin(), host_int_Parralel_S.end(), rand);

		cout << "Parralel size: " << host_int_Parralel_S.size() << endl;

		thrust::device_vector<char> d_vec_S = host_int_Parralel_S;

		high_resolution_clock::time_point start_S = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::stable_sort(d_vec_S.begin(), d_vec_S.end());
		high_resolution_clock::time_point stop_S = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec_S.begin(), d_vec_S.end(), host_int_Parralel_S.begin());

		auto durationParralel_S = duration_cast <milliseconds> (stop_S - start_S).count();
		cout << "Time: " << durationParralel_S << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		//////thrust parralel
		cout << "Parralel: Radix sort" << endl;

		thrust::host_vector<char> host_int_Parralel(N);
		std::generate(host_int_Parralel.begin(), host_int_Parralel.end(), rand);

		cout << "Parralel size: " << host_int_Parralel.size() << endl;
		//cout << host_int_Parralel.data() << endl;

		thrust::device_vector<char> d_vec = host_int_Parralel;

		high_resolution_clock::time_point start = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::sort(d_vec.begin(), d_vec.end());
		high_resolution_clock::time_point stop = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec.begin(), d_vec.end(), host_int_Parralel.begin());
		//cout << host_int_Parralel.data() << endl;

		auto durationParralel = duration_cast <milliseconds> (stop - start).count();
		cout << "Time: " << durationParralel << " milliseconds" << endl;

		cout << endl;
		cout << endl;
	
		
		string t(host_int_Parralel.begin(), host_int_Parralel.end());
		vector<string> tmp;
		tmp.push_back(t);

		

		/*countingRadixSort(tmp, N);*/

		//cout << "Serial Radix sort" << endl;

		//thrust::host_vector<char> host_int_Serial(N);
		//std::generate(host_int_Serial.begin(), host_int_Serial.end(), rand);

		//cout << "Serial size: " << host_int_Serial.size() << endl;

		////thrust::device_vector<unsigned long int> d_vec_S;
		//char* da = host_int_Serial.data();
		high_resolution_clock::time_point startS = high_resolution_clock::now();
		//// sort data on the device (846M keys per second on GeForce GTX 480)
		//serialRadixSort<char>(da, N);
		radixSort2(tmp, (int)(sizeof(tmp) / sizeof(tmp[0])));

		high_resolution_clock::time_point stopS = high_resolution_clock::now();

		auto durationS = duration_cast <milliseconds> (stopS - startS).count();
		cout << "Time: " << durationS << " milliseconds" << endl;

		//cout << endl;
		//cout << endl;
		break;
	}
	case 3:
	{
		cout << endl;
		cout << endl;
		cout << "Long" << endl << endl;

		cout << "Parralel:  Stable Radix sort" << endl;

		thrust::host_vector<long> host_int_Parralel_S(N);
		std::generate(host_int_Parralel_S.begin(), host_int_Parralel_S.end(), rand);

		cout << "Parralel size: " << host_int_Parralel_S.size() << endl;

		thrust::device_vector<long> d_vec_S = host_int_Parralel_S;

		high_resolution_clock::time_point start_S = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::stable_sort(d_vec_S.begin(), d_vec_S.end());
		high_resolution_clock::time_point stop_S = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec_S.begin(), d_vec_S.end(), host_int_Parralel_S.begin());

		auto durationParralel_S = duration_cast <milliseconds> (stop_S - start_S).count();
		cout << "Time: " << durationParralel_S << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		//////thrust parralel
		cout << "Parralel: Radix sort" << endl;

		thrust::host_vector<long> host_int_Parralel(N);
		std::generate(host_int_Parralel.begin(), host_int_Parralel.end(), rand);

		cout << "Parralel size: " << host_int_Parralel.size() << endl;

		thrust::device_vector<long> d_vec = host_int_Parralel;

		high_resolution_clock::time_point start = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::sort(d_vec.begin(), d_vec.end());
		high_resolution_clock::time_point stop = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec.begin(), d_vec.end(), host_int_Parralel.begin());

		auto durationParralel = duration_cast <milliseconds> (stop - start).count();
		cout << "Time: " << durationParralel << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		cout << "Serial Radix sort" << endl;

		thrust::host_vector<long> host_int_Serial(N);
		std::generate(host_int_Serial.begin(), host_int_Serial.end(), rand);

		cout << "Serial size: " << host_int_Serial.size() << endl;

		//thrust::device_vector<unsigned long int> d_vec_S;
		long* da = host_int_Serial.data();
		high_resolution_clock::time_point startS = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		serialRadixSort<long>(da, N);
		high_resolution_clock::time_point stopS = high_resolution_clock::now();

		auto durationS = duration_cast <milliseconds> (stopS - startS).count();
		cout << "Time: " << durationS << " milliseconds" << endl;

		cout << endl;
		cout << endl;
		break;
	}
	case 4:
	{
		cout << endl;
		cout << endl;
		cout << "float" << endl << endl;

		cout << "Parralel:  Stable Radix sort" << endl;


		thrust::host_vector<float> host_int_Parralel_S(N);
		std::generate(host_int_Parralel_S.begin(), host_int_Parralel_S.end(), rand);

		cout << "Parralel size: " << host_int_Parralel_S.size() << endl;

		thrust::device_vector<float> d_vec_S = host_int_Parralel_S;

		high_resolution_clock::time_point start_S = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::stable_sort(d_vec_S.begin(), d_vec_S.end());
		high_resolution_clock::time_point stop_S = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec_S.begin(), d_vec_S.end(), host_int_Parralel_S.begin());

		auto durationParralel_S = duration_cast <milliseconds> (stop_S - start_S).count();
		cout << "Time: " << durationParralel_S << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		//////thrust parralel
		cout << "Parralel: Radix sort" << endl;

		thrust::host_vector<float> host_int_Parralel(N);
		std::generate(host_int_Parralel.begin(), host_int_Parralel.end(), rand);

		cout << "Parralel size: " << host_int_Parralel.size() << endl;

		thrust::device_vector<float> d_vec = host_int_Parralel;

		high_resolution_clock::time_point start = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::sort(d_vec.begin(), d_vec.end());
		high_resolution_clock::time_point stop = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec.begin(), d_vec.end(), host_int_Parralel.begin());

		auto durationParralel = duration_cast <milliseconds> (stop - start).count();
		cout << "Time: " << durationParralel << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		cout << "Serial Radix sort" << endl;

		thrust::host_vector<float> host_int_Serial(N);
		thrust::host_vector<unsigned int> host_int_Serial_Int(0);
		std::generate(host_int_Serial.begin(), host_int_Serial.end(), rand);
		for each (float var in host_int_Serial)
		{
			host_int_Serial_Int.push_back(var);
			//cout << var << endl;
		}
		cout << host_int_Serial_Int.size() << endl;

		cout << "Serial size: " << host_int_Serial.size() << endl;

		//thrust::device_vector<unsigned long int> d_vec_S;
		unsigned int* da = host_int_Serial_Int.data();
		high_resolution_clock::time_point startS = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		serialRadixSort<unsigned int>(da, N);
		high_resolution_clock::time_point stopS = high_resolution_clock::now();

		auto durationS = duration_cast <milliseconds> (stopS - startS).count();
		cout << "Time: " << durationS << " milliseconds" << endl;

		cout << endl;
		cout << endl;
		break;
	}
	case 5:
	{
		cout << endl;
		cout << endl;
		cout << "double" << endl << endl;

		cout << "Parralel:  Stable Radix sort" << endl;

		thrust::host_vector<double> host_int_Parralel_S(N);
		std::generate(host_int_Parralel_S.begin(), host_int_Parralel_S.end(), rand);

		cout << "Parralel size: " << host_int_Parralel_S.size() << endl;

		thrust::device_vector<double> d_vec_S = host_int_Parralel_S;

		high_resolution_clock::time_point start_S = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::stable_sort(d_vec_S.begin(), d_vec_S.end());
		high_resolution_clock::time_point stop_S = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec_S.begin(), d_vec_S.end(), host_int_Parralel_S.begin());

		auto durationParralel_S = duration_cast <milliseconds> (stop_S - start_S).count();
		cout << "Time: " << durationParralel_S << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		//////thrust parralel
		cout << "Parralel: Radix sort" << endl;

		thrust::host_vector<double> host_int_Parralel(N);
		std::generate(host_int_Parralel.begin(), host_int_Parralel.end(), rand);

		cout << "Parralel size: " << host_int_Parralel.size() << endl;

		thrust::device_vector<double> d_vec = host_int_Parralel;

		high_resolution_clock::time_point start = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		thrust::sort(d_vec.begin(), d_vec.end());
		high_resolution_clock::time_point stop = high_resolution_clock::now();

		// transfer data back to host
		thrust::copy(d_vec.begin(), d_vec.end(), host_int_Parralel.begin());

		auto durationParralel = duration_cast <milliseconds> (stop - start).count();
		cout << "Time: " << durationParralel << " milliseconds" << endl;

		cout << endl;
		cout << endl;

		cout << "Serial Radix sort" << endl;

		thrust::host_vector<double> host_int_Serial(N);
		thrust::host_vector<unsigned int> host_int_Serial_IntD(0);
		std::generate(host_int_Serial.begin(), host_int_Serial.end(), rand);

		cout << "Serial size: " << host_int_Serial.size() << endl;
		for each (double var in host_int_Serial)
		{
			host_int_Serial_IntD.push_back(var);
			//cout << var << endl;
		}

		//thrust::device_vector<unsigned long int> d_vec_S;
		unsigned int* da = host_int_Serial_IntD.data();
		high_resolution_clock::time_point startS = high_resolution_clock::now();
		// sort data on the device (846M keys per second on GeForce GTX 480)
		serialRadixSort<unsigned int>(da, N);
		high_resolution_clock::time_point stopS = high_resolution_clock::now();

		auto durationS = duration_cast <milliseconds> (stopS - startS).count();
		cout << "Time: " << durationS << " milliseconds" << endl;

		cout << endl;
		cout << endl;
		break;
	}
	default:
		
		break;
	}

	
	
	 //sort data on the device (846M keys per second on GeForce GTX 480)
	//thrust::sort(thrust::host, d_vec_S.begin(), d_vec_S.end());
	
	
	 //transfer data back to host
	//thrust::copy(d_vec_S.begin(), d_vec_S.end(), host_int_Serial.begin());
//	auto durationSerial = duration_cast <milliseconds> (stopS - startS).count();
	/*for (int i = 0; i < N; i++) {
		if (da[i] > da[i + 1]) {
			printf("sort error at, hdata[%d] = %d, hdata[%d] = %d\n", i, da[i], i + 1, da[i + 1]);
			return 1;
		}
	}*/



	

	 
	

}





//
//#define BIT 32
//#define SIZE (33 * 1024)
//
///*Serial radix sort*/



//__device__ void radixSortParralel(unsigned int* sort_tmp, const int num_lists, const int num_element, const int tid, unsigned int* sort_tmp_1) {
//	//if (sort_tmp == NULL) {
//	//	printf("\nsort_tmp is null");
//	//}
//
//	for (int bit = 0; bit < BIT; bit++) {
//		const int bit_mask = (1 << bit);
//		int base_cnt_0 = 0, base_cnt_1 = 0;
//		//printf("\ntest1 + %d",tid );
//		for (int i = 0; i < num_element; i += num_lists) {
//			int elem = sort_tmp[i + tid];
//
//			//printf("\ntest %d + %d bit %d",tid, elem, bit_mask);
//			if ((elem & bit_mask) > 0) {
//				sort_tmp_1[base_cnt_1 + tid] = elem;
//				base_cnt_1 += num_lists;
//				//printf("\n element 1 %d = %d",i, elem);
//			}
//			else {
//				sort_tmp[base_cnt_0 + tid] = elem;
//				base_cnt_0 += num_lists;
//				//printf("\n element 0 %d = %d",i, elem);
//			}
//		}
//		//copy data back to source - first the zero list
//		for (int i = 0; i < base_cnt_1; i += num_lists) {
//			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
//			//printf("\nsort tmp: %d", sort_tmp_0[i + tid]);
//		}
//	}
//	__syncthreads();
//
//	//for (int i = 0; i < num_element; i++) {
//	//	printf("\n%d", sort_tmp[i]);
//	//}
//}
//int find_min(unsigned int * const src_array,
//	int * const list_indexes,
//	const int num_lists,
//	const int num_elements_per_list)
//{
//	int min_val = 0xFFFFFFFF;
//	int min_idx = 0;
//	// Iterate over each of the lists
//	for (int i = 0; i < num_lists; i++)
//	{
//		// If the current list has already been emptied
//		// then ignore it
//		if (list_indexes[i] < num_elements_per_list)
//		{
//			const int src_idx = i + (list_indexes[i] * num_lists);
//			const int data = src_array[src_idx];
//			if (data <= min_val)
//			{
//				min_val = data;
//				min_idx = i;
//			}
//		}
//	}
//	list_indexes[min_idx]++;
//	return min_val;
//}
//__device__ void copy_data_to_shared(unsigned int * data,
//	unsigned int * sort_tmp,
//	const int num_lists,
//	const int num_elements,
//	const int tid)
//{
//	// Copy data into temp store
//	for (int i = 0; i < num_elements; i += num_lists)
//	{
//		sort_tmp[i + tid] = data[i + tid];
//	}
//	__syncthreads();
//}
//__device__ void merge_array6(unsigned int * src_array,
//	unsigned int * dest_array,
//	const int num_lists,
//	const int num_elements,
//	const int tid)
//{
//	const int num_elements_per_list = (num_elements / num_lists);
//	
//	__shared__ int list_indexes[32];
//
//	list_indexes[tid] = 0;
//	// Wait for list_indexes[tid] to be cleared
//	__syncthreads();
//	// Iterate over all elements
//	for (int i = 0; i < num_elements; i++)
//	{
//		// Create a value shared with the other threads
//		__shared__ int min_val;
//		__shared__ int min_tid;
//		// Use a temp register for work purposes
//		int data;
//		// If the current list has not already been
//		// emptied then read from it, else ignore it
//		if (list_indexes[tid] < num_elements_per_list)
//		{
//			// Work out from the list_index, the index into
//			// the linear array
//			const int src_idx = tid + (list_indexes[tid] * num_lists);
//			// Read the data from the list for the given
//			// thread
//			data = src_array[src_idx];
//		}
//		else
//		{
//			data = 0xFFFFFFFF;
//		}
//		// Have thread zero clear the min values
//		if (tid == 0)
//		{
//			// Write a very large value so the first
//			// thread thread wins the min
//			min_val = 0xFFFFFFFF;
//			min_tid = 0xFFFFFFFF;
//		}
//		// Wait for all threads
//		__syncthreads();
//		// Have every thread try to store it�s value into
//		// min_val. Only the thread with the lowest value
//		// will win
//
//		atomicMin(&min_val, data);
//
//		// Make sure all threads have taken their turn.
//		__syncthreads();
//		// If this thread was the one with the minimum
//		if (min_val == data)
//		{
//			// Check for equal values
//			// Lowest tid wins and does the write
//
//			atomicMin(&min_tid, tid);
//		}
//		// Make sure all threads have taken their turn.
//		__syncthreads();
//		// If this thread has the lowest tid
//		if (tid == min_tid)
//		{
//			// Incremene the list pointer for this thread
//			list_indexes[tid]++;
//			// Store the winning value
//			dest_array[i] = data;
//		}
//	}
//}
//
//__global__ void gpu_sort_array_array(unsigned int * data, const int num_lists, const int num_elements)
//{
//	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
//
//	__shared__ unsigned int sort_tmp[64];
//	__shared__ unsigned int sort_tmp_1[64];
//
//	copy_data_to_shared(data, sort_tmp, num_lists,
//		num_elements, tid);
//	radixSortParralel(sort_tmp, num_lists, num_elements,
//		tid, sort_tmp_1);
//	merge_array6(sort_tmp, data, num_lists,
//		num_elements, tid);
//
//}
//
//void print(unsigned int* arr, int n)
//{
//	for (int i = 0; i < n; i++)
//		cout << (double)arr[i] / 100 << endl;
//}
//
//void menu() {
//	cout << "Izberi: " << endl;
//	cout << " 0 Exit " << endl;
//	cout << " 1 Serial sort" << endl;
//	cout << " 2 Parrallel sort" << endl;
//	cout << " 3 Thrust Parrallel sort" << endl;
//}
//
////void WriteFile(vector<double> vector) {
////	ofstream myfile;
////	myfile.open("out.txt");
////	for each (double var in vector)
////	{
////		myfile << var << " ";
////	}
////	myfile.close();
////}
//
//vector<unsigned int> ReadFile(string filepath) {
//	vector<unsigned int> vector;
//	ifstream inputFile(filepath);
//	double value;
//
//	//read the elements in the file into a vector  
//	while (inputFile >> value) {
//		vector.push_back(value * 100);
//	}
//	inputFile.close();
//	return vector;
//}
//
//int main(int argc, char * argv[]) {
//
//	int blockSize = 32;
//	int numBlocks = (SIZE + blockSize - 1) / blockSize;
//
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	float totalTime = 0;
//
//	std::ifstream fin(argv[1]);
//	//std::vector<unsigned int> host_double(1);
//
//	unsigned int host_data[SIZE];
//
//	// generate 32M random numbers serially
//	
//
//	if (!fin) {
//		cerr << "Datoteka ne obstaja " << argv[1] << endl;
//	}
//	else {
//		cout << "Datoteka obstaja " << argv[1] << endl;
//		//host_double = ReadFile(argv[1]);
//
//		//WriteFile(host_double);
//	}
//	
//
//	while (true) {
//		thrust::host_vector<unsigned int> host_double(1 << 20);
//		std::generate(host_double.begin(), host_double.end(), rand);
//		int size = host_double.size();
//
//		int izbiraAlg;
//		menu();
//		cin >> izbiraAlg;
//		if (izbiraAlg == 0) {
//			return 0;
//		}
//		else if (izbiraAlg == 1) {
//
//			//Serial radix sort
//
//			unsigned int* da = host_double.data();
//			int n = sizeof(host_double) / sizeof(host_double[0]);
//			//auto t1 = high_resolution_clock::now();
//			high_resolution_clock::time_point start = high_resolution_clock::now();
//			serialRadixSort(da, size);
//			high_resolution_clock::time_point stop = high_resolution_clock::now();
//			auto duration = duration_cast <milliseconds> (stop - start).count();
//			cout << endl << "Serial radix sort " << endl;
//			//print(da, size);
//			for (int i = 0; i < size - 1; i++) {
//				if (host_double[i] > host_double[i + 1]) {
//					printf("sort error at, hdata[%d] = %d, hdata[%d] = %d\n", i, host_double[i], i + 1, host_double[i + 1]);
//					return 1;
//				}
//			}
//			cout << "Total time: " << duration << "ms" << endl;
//			cout << "Success" << endl;
//
//		}
//		else if (izbiraAlg == 2) {
//
//			//Parrallel radix sort
//			cout << endl;
//			cout << "Parrallel sort " << endl;
//			std::copy(host_double.begin(), host_double.end(), host_data);
//			/*cout << "Not sorted!" << endl;
//			print(host_data, size);*/
//
//			unsigned int* ddata;
//			unsigned int* ddata_tmp;
//			//
//
//			//for (int lcount = 0; lcount < LOOPS; lcount++) {
//			//	unsigned int range = 1U << UPPER_BIT;
//			//	//for (int i = 0; i < SIZE; i++) host_data[i] = rand() % range;
//				//hipMemcpyToSymbol(HIP_SYMBOL(device_data), host_data, SIZE * sizeof(unsigned int));
//			hipMalloc((void**)&ddata_tmp, SIZE * sizeof(unsigned int));
//			hipMalloc((void**)&ddata, SIZE * sizeof(unsigned int));
//			hipMemcpy(ddata, host_data, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
//
//			high_resolution_clock::time_point start = high_resolution_clock::now();
//			//radixSortParralel << <1, 10 >> >(ddata, ddata_tmp);
//			//radixSort << <(numBlocks, blockSize) >> > (ddata, ddata_tmp);
//			gpu_sort_array_array << < 1, 2 >> > (ddata, 32, 64);
//			high_resolution_clock::time_point stop = high_resolution_clock::now();
//			host_data[SIZE];
//
//			hipMemcpy(host_data, ddata, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
//
//			//hipMemcpyFromSymbol(host_data, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));
//			auto durationParralel = duration_cast <milliseconds> (stop - start).count();
//			//	/*for (int i = 0; i < SIZE - 1; i++) {
//			//	if (host_data[i] > host_data[i + 1]) {
//			//	printf("sort error at loop %d, hdata[%d] = %d, hdata[%d] = %d\n", lcount, i, host_data[i], i + 1, host_data[i + 1]);
//			//	return 1;
//			//	}
//			//	}*/
//			//	cout << " Sorted data: " << endl;
//			//	for (int i = 0; i < size; i++) {
//			//		cout << i << "     " <<(double)host_data[i] / 100 << endl;
//			//	}
//			//	cout << " Time parralel sort: " << durationParralel << endl;
//			//}
//			cout << endl << "Sorted" << endl;
//			print(host_data, size);
//
//		}
//		else {
//			//thrust
//
//			cout << "Parralel Radix sort Thrust" << endl;
//
//			// transfer data to the device
//			thrust::device_vector<int> d_vec = host_double;
//
//			high_resolution_clock::time_point start = high_resolution_clock::now();
//			// sort data on the device (846M keys per second on GeForce GTX 480)
//			thrust::sort(d_vec.begin(), d_vec.end());
//			high_resolution_clock::time_point stop = high_resolution_clock::now();
//
//			// transfer data back to host
//			thrust::copy(d_vec.begin(), d_vec.end(), host_double.begin());
//
//			auto durationParralel = duration_cast <milliseconds> (stop - start).count();
//			cout << "Time: " << durationParralel << "ms" << endl;
//			/*for each (unsigned int var in host_double)
//			{
//				double tmp = (double)var / 100;
//				cout << tmp << endl;
//			}*/
//		}
//	}
//
//
//	return 0;
//}


