#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>
#include <bitset>
//
//#define SIZE 5
//#define BIT 5
//
using namespace std;
//
//__device__ unsigned int device_data[SIZE];
//__device__ int ddata_s[SIZE];
//
////Test v gpu in nazaj
//__global__ void gpucopy(int* src, int* dst)
//{
//	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
//	dst[i] = src[i];
//}
//
//__global__ void radixSort(int* sort_tmp, const int num_lists, const int num_element, int* sort_tmp_1) {
//	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
//	//printf("\n%d num list: %d" ,tid, num_lists );
//	//printf("\n num element: %d" , num_element);
//	if (sort_tmp == NULL) {
//		printf("\nsort_tmp is null");
//	}
//	
//	
//	for (int bit = 0; bit < BIT; bit++) {
//		int base_cnt_0 = 0, base_cnt_1 = 0;
//		//printf("\ntest1 + %d",tid );
//		for (int i = 0; i < num_element; i+=num_lists) {
//			int elem = sort_tmp[i + tid];
//			const int bit_mask = (1 << bit);
//			//printf("\ntest %d + %d bit %d",tid, elem, bit_mask);
//			if ((elem & bit_mask) > 0) {
//				sort_tmp_1[base_cnt_1 + tid] = elem;
//				base_cnt_1 += num_lists;
//				//printf("\n element 1 %d = %d",i, elem);
//			}
//			else {
//				sort_tmp[base_cnt_0 + tid] = elem;
//				base_cnt_0+=num_lists;
//				//printf("\n element 0 %d = %d",i, elem);
//			}
//		}
//		//copy data back to source - first the zero list
//		for (int i = 0; i < base_cnt_1; i += num_lists) {
//			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
//			//printf("\nsort tmp: %d", sort_tmp_0[i + tid]);
//		}
//	}
//	
//	__syncthreads();
//
//	for (int i = 0; i < num_element; i++) {
//		printf("\n%d", sort_tmp[i]);
//	}
//}
//
//
//void WriteFile(vector<double> vector) {
//	ofstream myfile;
//	myfile.open("out.txt");
//	for each (double var in vector)
//	{
//		myfile << var << " ";
//	}
//	myfile.close();
//}
vector<unsigned int> ReadFile(string filepath) {
	vector<unsigned int> vector;
	ifstream inputFile(filepath);
	double value;

	// read the elements in the file into a vector  
	while (inputFile >> value) {
		//string binary = bitset<16>(value * 100).to_string();
		vector.push_back(value * 100);
	}
	for each (unsigned int var in vector)
	{
		cout << var << " " << endl;
	}
	inputFile.close();
	return vector;
}
//
//int main(int argc, char * argv[])
//{
//	cout << argv[1] << endl;
//
//	std::ifstream fin(argv[1]);
//	
//	std::vector<string> host_double(7);
//	std::vector<int> host_double2(7);
//
//	float totalTime = 0;
//	float milliseconds = 0;
//
//	//Time
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//
//	if (!fin) {
//		cerr << "Datoteka ne obstaja " << argv[1] << endl;
//	}
//	else {
//		cout << "Datoteka obstaja " << argv[1] << endl;
//		host_double = ReadFile(argv[1]);
//
//		//WriteFile(host_double);
//	}
//	int size = host_double.size();
//	
//	int* device = NULL;
//	int* tmp_0 = NULL;
//	int* tmp_1 = NULL;
//	
//	hipMalloc((void**)&device, size * sizeof(int));
//	hipMalloc((void**)&tmp_0, size * sizeof(int));
//	hipMalloc((void**)&tmp_1, size * sizeof(int));
//
//	string s = host_double[0];
//	cout << "test: " << s[0] << endl;
//	
//	
//	//string binary = bitset<16>(222).to_string(); //to binary
//	//cout << binary << "\n";	
//
//	//unsigned long decimal = bitset<16>(binary).to_ulong();
//	//cout << decimal << "\n";
//
//	//num_list 32 
//
//	hipMemcpy(device,&*host_double.data(), size * sizeof(int), hipMemcpyHostToDevice);
//	
//	//radixSort << < 1,5>> >(device, SIZE, size, tmp_1);
//	//ParalelRadixSort << <(size + size - 1) / size, size >> >();
//	//gpucopy << <1,SIZE>> >(device, device2);
//	
//	hipDeviceSynchronize();
//
//	hipMemcpy(&*host_double.data(), device, size * sizeof(int), hipMemcpyDeviceToHost);
//
//	//cout << endl << "host data" << endl;
//	//for each (string var in host_double)
//	//{
//	//	double d = (double)var / 100;
//	//	cout << d << " " << endl;
//	//}
//
//	hipFree(device);
//	hipFree(tmp_0);
//	hipFree(tmp_1);
//
//	//std::vector<int> v(10);
//
//	//unsigned int hdata[SIZE];
//
//
//	//	int size = host_double.size();
//	//	//vector.resize(size);
//
//	//	std::copy(host_double.begin(), host_double.end(), hdata);
//	//	//int *a = vector.data();
//
//	//	// Copy data from host to device
//	//	hipMemcpyToSymbol(HIP_SYMBOL(device_data), hdata, SIZE * sizeof(unsigned int));
//
//	//	// Execution time measurement, that point starts the clock
//	//	hipEventRecord(start);
//	//	ParalelRadixSort << <1, SIZE>> >();
//
//	//	// Execution time measurement, that point stops the clock
//	//	hipEventRecord(stop);
//
//	//	// Make kernel function synchronous
//	//	hipDeviceSynchronize();
//
//	//	hipEventElapsedTime(&milliseconds, start, stop);
//	//	totalTime += milliseconds;
//	//	//milliseconds = 0;
//
//	//	// Copy data from device to host
//	//	hipMemcpyFromSymbol(hdata, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));
//
//	//	/*std::vector<int> v(hdata, hdata + sizeof hdata / sizeof hdata[0]);
//	//	v.resize(size);*/
//	//	milliseconds = 0;
//	//
//
//	///*printf("TIME %4.2fs", milliseconds);
//	//printf("\n");
//	//printf("Effective Bandwidth (GB/s): %fn", SIZE * 4 * 3 / milliseconds / 1e6);
//	//printf("\n");*/
//	//cout << endl;
//	//cout << "Host Data" << endl;
//	///*for each (int var in v)
//	//{
//	//	double d = (double)var / 100;
//	//	cout << d << " " << endl;
//	//}*/
//	//for (int i = 0; i < SIZE; i++) {
//	//	double d = (double)hdata[i] / 100;
//	//	cout << d << endl;
//	//}
//	//
//	//cout << endl;
//	//cout << "TIME %4.2fs " << totalTime << endl;
//
//    return 0;
//}
//


#define WSIZE 32
#define LOOPS 1
#define UPPER_BIT 31
#define LOWER_BIT 0

__device__ unsigned int ddata[WSIZE];

// naive warp-level bitwise radix sort

__global__ void mykernel() {
	__shared__ volatile unsigned int sdata[WSIZE * 2];
	// load from global into shared variable
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	sdata[tid] = ddata[tid];
	unsigned int bitmask = 1 << LOWER_BIT;
	unsigned int offset = 0;
	unsigned int thrmask = 0xFFFFFFFFU << tid;
	unsigned int mypos;
	//  for each LSB to MSB
	for (int i = LOWER_BIT; i <= UPPER_BIT; i++) {
		unsigned int mydata = sdata[((WSIZE - 1) - tid) + offset];
		unsigned int mybit = mydata&bitmask;
		// get population of ones and zeroes (cc 2.0 ballot)
		unsigned int ones = __ballot(mybit); // cc 2.0
		unsigned int zeroes = ~ones;
		offset ^= WSIZE; // switch ping-pong buffers
						 // do zeroes, then ones
		if (!mybit) // threads with a zero bit
					// get my position in ping-pong buffer
			mypos = __popc(zeroes&thrmask);
		else        // threads with a one bit
					// get my position in ping-pong buffer
			mypos = __popc(zeroes) + __popc(ones&thrmask);
		// move to buffer  (or use shfl for cc 3.0)
		sdata[mypos - 1 + offset] = mydata;
		// repeat for next bit
		bitmask <<= 1;
	}
	// save results to global
	ddata[tid] = sdata[tid + offset];
}

int main(int argc, char * argv[]) {
	std::ifstream fin(argv[1]);
	std::vector<unsigned int> host_double(WSIZE);

	unsigned int hdata[WSIZE];

	

	//if (!fin) {
	//	cerr << "Datoteka ne obstaja " << argv[1] << endl;
	//}
	//else {
	//	cout << "Datoteka obstaja " << argv[1] << endl;
	//	host_double = ReadFile(argv[1]);
	//	
	//	//WriteFile(host_double);
	//}
	//std::copy(host_double.begin(), host_double.end(), hdata);

	for (int lcount = 0; lcount < LOOPS; lcount++) {
		unsigned int range = 1U << UPPER_BIT;
		for (int i = 0; i < WSIZE; i++) hdata[i] = rand() % range;
		hipMemcpyToSymbol(HIP_SYMBOL(ddata), hdata, WSIZE * sizeof(unsigned int));
		mykernel << <1, 32 >> >();
		hipMemcpyFromSymbol(hdata, HIP_SYMBOL(ddata), WSIZE * sizeof(unsigned int));
		for (int i = 0; i < WSIZE - 1; i++) if (hdata[i] > hdata[i + 1]) { printf("sort error at loop %d, hdata[%d] = %d, hdata[%d] = %d\n", lcount, i, hdata[i], i + 1, hdata[i + 1]); return 1; }
		 printf("sorted data:\n");
		for (int i = 0; i < WSIZE; i++) cout << (double)hdata[i] / 100 << endl;
	}
	printf("Success!\n");
	return 0;
}
