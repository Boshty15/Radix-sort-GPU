#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>
#include <bitset>
#include <time.h>
#include <chrono>

//#define SIZE 32
//#define BIT 32
//
using namespace std;
using namespace chrono;
//
//__device__ unsigned int device_data[SIZE];
//__device__ int ddata_s[SIZE];
//
////Test v gpu in nazaj
//__global__ void gpucopy(int* src, int* dst)
//{
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	int stride = blockDim.x * gridDim.x;
//	for (int i = index; i < SIZE; i += stride) {
//		if(src[i] > 50)
//			src[i] = dst[i] + 100;
//		else 
//			src[i] = dst[i] - 100;
//	}
//		
//}
//
//__global__ void radixSort(int* sort_tmp, const int num_lists, const int num_element, int* sort_tmp_1) {
//	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
//	int stride = blockDim.x * gridDim.x;
//	//printf("\n%d num list: %d" ,tid, num_lists );
//	//printf("\n num element: %d" , num_element);
//	if (sort_tmp == NULL) {
//		printf("\nsort_tmp is null");
//	}
//	
//	
//	//for (int bit = 0; bit < BIT; bit++) {
//		int base_cnt_0 = 0, base_cnt_1 = 0;
//		//printf("\ntest1 + %d",tid );
//		for (int i = tid; i < num_element; i+=stride) {
//			int elem = sort_tmp[i];
//			const int bit_mask = (1 << i);
//			//printf("\ntest %d + %d bit %d",tid, elem, bit_mask);
//			if ((elem & bit_mask) > 0) {
//				sort_tmp_1[base_cnt_1] = elem;
//				base_cnt_1 += num_lists;
//				//printf("\n element 1 %d = %d",i, elem);
//			}
//			else {
//				sort_tmp[base_cnt_0] = elem;
//				base_cnt_0+=num_lists;
//				//printf("\n element 0 %d = %d",i, elem);
//			}
//		}
//		//copy data back to source - first the zero list
//		for (int i = 0; i < base_cnt_1; i += num_lists) {
//			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
//			//printf("\nsort tmp: %d", sort_tmp_0[i + tid]);
//		}
//	//}
//	
//	__syncthreads();
//
//	for (int i = 0; i < num_element; i++) {
//		printf("\n%d", sort_tmp[i]);
//	}
//}
//
//
//void WriteFile(vector<double> vector) {
//	ofstream myfile;
//	myfile.open("out.txt");
//	for each (double var in vector)
//	{
//		myfile << var << " ";
//	}
//	myfile.close();
//}
vector<unsigned int> ReadFile(string filepath) {
	vector<unsigned int> vector;
	ifstream inputFile(filepath);
	double value;

	 //read the elements in the file into a vector  
	while (inputFile >> value) {
		vector.push_back(value * 100);
	}
	/*for each (unsigned int var in vector)
	{
		cout << var << " " << endl;
	}*/
	inputFile.close();
	return vector;
}
//
//int main(int argc, char * argv[])
//{
//	cout << argv[1] << endl;
//
//	std::ifstream fin(argv[1]);
//	
//	std::vector<unsigned int> host_double(7);
//	std::vector<int> host_double2(7);
//
//	float totalTime = 0;
//	float milliseconds = 0;
//
//	//Time
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//
//	if (!fin) {
//		cerr << "Datoteka ne obstaja " << argv[1] << endl;
//	}
//	else {
//		cout << "Datoteka obstaja " << argv[1] << endl;
//		host_double = ReadFile(argv[1]);
//
//		//WriteFile(host_double);
//	}
//	int size = host_double.size();
//	
//	int* device = NULL;
//	int* tmp_0 = NULL;
//	int* tmp_1 = NULL;
//	
//	hipMalloc((void**)&device, size * sizeof(int));
//	hipMalloc((void**)&tmp_0, size * sizeof(int));
//	hipMalloc((void**)&tmp_1, size * sizeof(int));
//
//	int blockSize = 4;
//	int numBlocks = (SIZE + blockSize - 1) / blockSize;
//	
//	//string binary = bitset<16>(222).to_string(); //to binary
//	//cout << binary << "\n";	
//
//	//unsigned long decimal = bitset<16>(binary).to_ulong();
//	//cout << decimal << "\n";
//
//	//num_list 32 
//
//	hipMemcpy(device,&*host_double.data(), size * sizeof(int), hipMemcpyHostToDevice);
//	
//	//radixSort << <numBlocks, blockSize>> >(device, SIZE, size, tmp_1);
//	//ParalelRadixSort << <(size + size - 1) / size, size >> >();
//	gpucopy << <numBlocks, blockSize>> >(device, device);
//	
//	hipDeviceSynchronize();
//
//	hipMemcpy(&*host_double.data(), device, size * sizeof(int), hipMemcpyDeviceToHost);
//
//	cout << endl << "host data" << endl;
//	for each (unsigned int var in host_double)
//	{
//		double d = (double)var / 100;
//		cout << d << " " << endl;
//	}
//
//	hipFree(device);
//	hipFree(tmp_0);
//	hipFree(tmp_1);
//
//	//std::vector<int> v(10);
//
//	//unsigned int hdata[SIZE];
//
//
//	//	int size = host_double.size();
//	//	//vector.resize(size);
//
//	//	std::copy(host_double.begin(), host_double.end(), hdata);
//	//	//int *a = vector.data();
//
//	//	// Copy data from host to device
//	//	hipMemcpyToSymbol(HIP_SYMBOL(device_data), hdata, SIZE * sizeof(unsigned int));
//
//	//	// Execution time measurement, that point starts the clock
//	//	hipEventRecord(start);
//	//	ParalelRadixSort << <1, SIZE>> >();
//
//	//	// Execution time measurement, that point stops the clock
//	//	hipEventRecord(stop);
//
//	//	// Make kernel function synchronous
//	//	hipDeviceSynchronize();
//
//	//	hipEventElapsedTime(&milliseconds, start, stop);
//	//	totalTime += milliseconds;
//	//	//milliseconds = 0;
//
//	//	// Copy data from device to host
//	//	hipMemcpyFromSymbol(hdata, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));
//
//	//	/*std::vector<int> v(hdata, hdata + sizeof hdata / sizeof hdata[0]);
//	//	v.resize(size);*/
//	//	milliseconds = 0;
//	//
//
//	///*printf("TIME %4.2fs", milliseconds);
//	//printf("\n");
//	//printf("Effective Bandwidth (GB/s): %fn", SIZE * 4 * 3 / milliseconds / 1e6);
//	//printf("\n");*/
//	//cout << endl;
//	//cout << "Host Data" << endl;
//	///*for each (int var in v)
//	//{
//	//	double d = (double)var / 100;
//	//	cout << d << " " << endl;
//	//}*/
//	//for (int i = 0; i < SIZE; i++) {
//	//	double d = (double)hdata[i] / 100;
//	//	cout << d << endl;
//	//}
//	//
//	//cout << endl;
//	//cout << "TIME %4.2fs " << totalTime << endl;
//
//    return 0;
//}



#define SIZE 100
#define LOOPS 1
#define UPPER_BIT 31
#define LOWER_BIT 0
int getMax(unsigned int* arr, int n)
{
	int mx = arr[0];
	for (int i = 1; i < n; i++)
		if (arr[i] > mx)
			mx = arr[i];
	return mx;
}

// A function to do counting sort of arr[] according to
// the digit represented by exp.
void countSort(unsigned int* arr, int n, int exp)
{
// output array
	vector<unsigned int> ve(n);
	unsigned int* output = ve.data();
	int i, count[10] = { 0 };

	// Store count of occurrences in count[]
	for (i = 0; i < n; i++)
		count[(arr[i] / exp) % 10]++;

	// Change count[i] so that count[i] now contains actual
	//  position of this digit in output[]
	for (i = 1; i < 10; i++)
		count[i] += count[i - 1];

	// Build the output array
	for (i = n - 1; i >= 0; i--)
	{
		output[count[(arr[i] / exp) % 10] - 1] = arr[i];
		count[(arr[i] / exp) % 10]--;
	}

	// Copy the output array to arr[], so that arr[] now
	// contains sorted numbers according to current digit
	for (i = 0; i < n; i++)
		arr[i] = output[i];
}

// The main function to that sorts arr[] of size n using 
// Radix Sort
void serialRadixSort(unsigned int* arr, int n)
{
	// Find the maximum number to know number of digits
	int m = getMax(arr, n);

	// Do counting sort for every digit. Note that instead
	// of passing digit number, exp is passed. exp is 10^i
	// where i is current digit number
	for (int exp = 1; m / exp > 0; exp *= 10)
		countSort(arr, n, exp);
}

// A utility function to print an array


__device__ unsigned int device_data[SIZE];

// naive warp-level bitwise radix sort

__global__ void radixSort() {
	__shared__ volatile unsigned int shared_data[SIZE * 2];
	// load from global into shared variable
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	shared_data[tid] = device_data[tid];
	unsigned int bitmask = 1 << LOWER_BIT;
	unsigned int offset = 0;
	unsigned int thrmask = 0xFFFFFFFFU << tid;
	unsigned int mypos;

	//  for each LSB to MSB
	for (int i = LOWER_BIT; i <= UPPER_BIT; i++) {
		/*unsigned int mydata = shared_data[((SIZE - 1) - tid) + offset];*/
		unsigned int mydata = shared_data[((SIZE - 1) - tid) + offset];
		unsigned int mybit = mydata&bitmask;
		// get population of ones and zeroes (cc 2.0 ballot)
		unsigned int ones = __ballot(mybit); // cc 2.0
		unsigned int zeroes = ~ones;
		offset ^= SIZE; // switch ping-pong buffers
						 // do zeroes, then ones
		if (!mybit) // threads with a zero bit
					// get my position in ping-pong buffer
			mypos = __popc(zeroes&thrmask);
		else        // threads with a one bit
					// get my position in ping-pong buffer
			mypos = __popc(zeroes) + __popc(ones&thrmask);
		// move to buffer  (or use shfl for cc 3.0)
		shared_data[mypos - 1 + offset] = mydata;
		// repeat for next bit
		bitmask <<= 1;
	}
	// save results to global
	device_data[tid] = shared_data[tid + offset];
	

}
void print(unsigned int* arr, int n)
{
	for (int i = 0; i < n; i++)
		cout << (double)arr[i] / 100 << endl;
}

int main(int argc, char * argv[]) {

	int blockSize = 128;
	int numBlocks = (SIZE + blockSize - 1) / blockSize;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float totalTime = 0;

	std::ifstream fin(argv[1]);
	std::vector<unsigned int> host_double(SIZE);

	unsigned int host_data[SIZE];

	if (!fin) {
		cerr << "Datoteka ne obstaja " << argv[1] << endl;
	}
	else {
		cout << "Datoteka obstaja " << argv[1] << endl;
		host_double = ReadFile(argv[1]);
		
		//WriteFile(host_double);
	}
	int size = host_double.size();

	//unsigned int* da = host_double.data();
	////int arr[] = { 170, 45, 75, 90, 802, 24, 2, 66, 170, 45, 75, 90, 802, 24, 2, 66 };
	//int n = sizeof(host_double) / sizeof(host_double[0]);
	//auto t1 = high_resolution_clock::now();
	//serialRadixSort(da, size);
	//auto t2 = high_resolution_clock::now();
	//auto diff = duration_cast<duration<double>>(t2 - t1);
	//// now elapsed time, in seconds, as a double can be found in diff.count()
	//long ms = (long)(1000 * diff.count());
	//cout << endl << "Serial radix sort " << endl;
	////print(da, size);
	//for (int i = 0; i < size - 1; i++) {
	//		if (host_double[i] > host_double[i + 1]) {
	//			printf("sort error at, hdata[%d] = %d, hdata[%d] = %d\n", i, host_double[i], i + 1, host_double[i + 1]);
	//			return 1; 
	//		}
	//	}
	//cout << "Total time: " << ms << "ms" << endl;
	//cout << "Success" << endl;

	std::copy(host_double.begin(), host_double.end(), host_data);

	for (int lcount = 0; lcount < LOOPS; lcount++) {
		unsigned int range = 1U << UPPER_BIT;
		//for (int i = 0; i < SIZE; i++) host_data[i] = rand() % range;
		hipMemcpyToSymbol(HIP_SYMBOL(device_data), host_data, SIZE *  sizeof(unsigned int));
		radixSort << <numBlocks,blockSize >> >();
		hipMemcpyFromSymbol(host_data, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));
	/*for (int i = 0; i < SIZE - 1; i++) {
			if (host_data[i] > host_data[i + 1]) { 
				printf("sort error at loop %d, hdata[%d] = %d, hdata[%d] = %d\n", lcount, i, host_data[i], i + 1, host_data[i + 1]);
				return 1; 
			}
		}*/
		printf("sorted data:\n");
		for (int i = 0; i < size; i++) {
			cout << (double)host_data[i] / 100 << endl;
		}
	}
	printf("Success!\n");
	return 0;
}
