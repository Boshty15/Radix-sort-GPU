#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>

#define SIZE 512

using namespace std;

__device__ unsigned int device_data[SIZE];

//Test v gpu in nazaj
__global__ void gpucopy(int* src, int* dst)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;;
	dst[i] = src[i];
}

__global__ void radix_sort(int* const sort_tmp, const int num_list, const int num_elements, const int tid, int* const sort_tmp_1) {
	//sort into num_list, lists
	//aplly radixx sort on 32 bits of data
	for (int bit = 0; bit < 32; bit++) {
		const unsigned int bit_mask = (1 << bit);
		unsigned int base_cnt_0 = 0, base_cnt_1 = 0;

		for (int i = 0; i < num_elements; i++) {
			const int element = sort_tmp[i + tid];

			if ((element & bit_mask) > 0) {
				sort_tmp_1[base_cnt_1 + tid] = element;
				base_cnt_1 += num_list;

			}
			else {
				sort_tmp[base_cnt_1 + tid] = element;
				base_cnt_0 += num_elements;
			}
		}
		//copy data back to source from the ones's list
		for (int i = 0; i < base_cnt_1; i++) {
			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
		}


	}
}

//__global__ void	ParalelRadixSort()
//{
//	//sprememba
//	__shared__ volatile unsigned int shared_data[SIZE * 2];
//
//	//spremena za commit
//	shared_data[threadIdx.x] = device_data[threadIdx.x];
//		
//	unsigned int bitmask = 1 << 0;
//	unsigned int offset = 0;
//	// -1, -2, -4, -8, -16, -32, -64, -128, -256,...
//	unsigned int thrmask = 0xFFFFFFFFU << threadIdx.x;
//	unsigned int mypos;
//
//	for (int i = 0; i <= 10; i++)
//	{
//		unsigned int mydata = shared_data[((SIZE - 1) - blockIdx.x * blockDim.x + threadIdx.x) + offset];
//		unsigned int mybit = mydata&bitmask;
//		// Get population of ones and zeroes
//		unsigned int ones = __ballot(mybit);
//		unsigned int zeroes = ~ones;
//		// Switch ping-pong buffers
//		offset ^= SIZE;
//
//		// Do zeroes, then ones
//		if (!mybit)
//		{
//			mypos = __popc(zeroes&thrmask);
//		}
//		else {      // Threads with a one bit
//					// Get my position in ping-pong buffer
//			mypos = __popc(zeroes) + __popc(ones&thrmask);
//		}
//
//		// Move to buffer  (or use shfl for cc 3.0)
//		shared_data[mypos - 1 + offset] = mydata;
//		// Repeat for next bit
//		bitmask <<= 1;
//
//		device_data[threadIdx.x] = shared_data[threadIdx.x + offset];
//	}
//		
//
//}

void WriteFile(vector<double> vector) {
	ofstream myfile;
	myfile.open("out.txt");
	for each (double var in vector)
	{
		myfile << var << " ";
	}
	myfile.close();
}
vector<int> ReadFile(string filepath) {
	vector<int> vector;
	ifstream inputFile(filepath);
	double value;

	// read the elements in the file into a vector  
	while (inputFile >> value) {
		vector.push_back(value * 10);
	}
	for each (int var in vector)
	{
		cout << var << " ";
	}
	inputFile.close();
	return vector;
}

int main(int argc, char * argv[])
{
	cout << argv[1] << endl;

	std::ifstream fin(argv[1]);
	
	std::vector<int> host_double(9);
	std::vector<int> host_double2(9);

	float totalTime = 0;
	float milliseconds = 0;

	//Time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (!fin) {
		cerr << "Datoteka ne obstaja " << argv[1] << endl;
	}
	else {
		cout << "Datoteka obstaja " << argv[1] << endl;
		host_double = ReadFile(argv[1]);

		//WriteFile(host_double);
	}
	int size = host_double.size();
	
	int* device = NULL;
	int* device2 = NULL;

	hipMalloc((void**)&device, size * sizeof(int));
	hipMalloc((void**)&device2, size * sizeof(int));

	//num_list 32 

	hipMemcpy(device, host_double.data(), size * sizeof(int), hipMemcpyHostToDevice);
	
	radix_sort << <(1, size) >> > ();
	//ParalelRadixSort << < 1, size >> >();
	//gpucopy << <(size + size - 1) / size, size >> >(device, device2);
	
	hipMemcpy(host_double2.data(), device2, size * sizeof(int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	cout << endl << "host data" << endl;
	for each (int var in host_double2)
	{
		cout << var << " ";
	}

	hipFree(device);

	//int size = vector.size();
	////vector.resize(size);

	//std::copy(vector.begin(), vector.end(), hdata);
	////int *a = vector.data();

	//// Copy data from host to device
	//hipMemcpyToSymbol(HIP_SYMBOL(device_data), hdata, SIZE * sizeof(unsigned int));

	//// Execution time measurement, that point starts the clock
	//hipEventRecord(start);
	//ParalelRadixSort << < 1, SIZE >> >();
	//	
	//// Execution time measurement, that point stops the clock
	//hipEventRecord(stop);

	//// Make kernel function synchronous
	//hipDeviceSynchronize();

	//hipEventElapsedTime(&milliseconds, start, stop);
	//totalTime += milliseconds;
	////milliseconds = 0;

	//// Copy data from device to host
	//hipMemcpyFromSymbol(hdata, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));

	//std::vector<int> v(hdata, hdata + sizeof hdata / sizeof hdata[0]);
	//v.resize(size);

	///*printf("TIME %4.2fs", milliseconds);
	//printf("\n");
	//printf("Effective Bandwidth (GB/s): %fn", SIZE * 4 * 3 / milliseconds / 1e6);
	//printf("\n");*/
	//cout << endl;
	//cout << "Host Data" << endl;
	//for each (int var in v)
	//{
	//	cout << var << " ";
	//}
	///*cout << "Device Data" << endl;
	//for each (int var in device_data)
	//{
	//	cout << var << " ";
	//}
	//*/
	//cout << endl;
	//cout << "TIME %4.2fs " << milliseconds << endl;

    return 0;
}







//
//#define nTPB 512
//
//__global__ void gpucopy(int* src, int* dst)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;;
//	dst[i] = src[i];
//}
//
//int main()
//{
//	const int arraySize = 500; // >= 1025 will fail on my system!
//
//	int* data1 = new int[arraySize];
//	int* data2 = new int[arraySize];
//	// Initialized both data1 and data2
//	// ... 
//	for (int i = 0; i < arraySize; i++) {
//		data1[i] = 2 * i;
//		//cout << data1[i] + " ";
//	}
//		
//
//	int* dev_data1 = NULL;
//	int* dev_data2 = NULL;
//	// Initialized both dev_data1 and dev_data2
//	// ... 
//	hipMalloc(&dev_data1, arraySize * sizeof(int));
//	hipMalloc(&dev_data2, arraySize * sizeof(int));
//
//	// copy data1 to device
//	hipMemcpy(dev_data1, data1, arraySize * sizeof(int), hipMemcpyHostToDevice);
//
//	//// copy dev_data1 to dev_data2 with gpu
//	//gpucopy << <1, arraySize >> >(dev_data1, dev_data2);
//	gpucopy << <(arraySize + nTPB - 1) / nTPB, nTPB >> >(dev_data1, dev_data2);
//
//	// copy dev_data2 to data
//	hipMemcpy(data2, dev_data2, arraySize * sizeof(int), hipMemcpyDeviceToHost);
//
//
//	for (int i = 0; i<arraySize; i++)
//		if (data2[i] != data1[i])
//			cout << "Error: data is different - data2[" << i << "] is " << data2[i] << endl;
//
//	return 0;
//}
//
//
//
