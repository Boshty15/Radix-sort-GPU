#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""


#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>

#define SIZE 32 //512
#define BIT

using namespace std;

__device__ unsigned int device_data[SIZE];
__device__ int ddata_s[SIZE];

//Test v gpu in nazaj
__global__ void gpucopy(int* src, int* dst)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;;
	dst[i] = src[i];
}

__global__ void radixSort(int* const sort_tmp, const int num_lists, const int num_element, int* const sort_tmp_0, int* const sort_tmp_1) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	for (int bit = 0; bit < 32; bit++) {
		int base_cnt_0 = 0, base_cnt_1 = 0;

		for (int i = 0; i < num_element; i++) {

		}
	}
}

__global__ void	ParalelRadixSort()
{
	int tib = (blockIdx.x * blockDim.x) + threadIdx.x;;
	//sprememba
	__shared__ volatile unsigned int shared_data[SIZE * 2];

	//spremena za commit
	shared_data[tib] = device_data[tib];
		
	unsigned int bitmask = 1 << 0;
	unsigned int offset = 0;
	// -1, -2, -4, -8, -16, -32, -64, -128, -256,...
	unsigned int thrmask = 0xFFFFFFFFU << tib;
	unsigned int mypos;

	for (int i = 0; i <= SIZE; i++){
		unsigned int mydata = shared_data[((SIZE - 1) - tib) + offset];
		unsigned int mybit = mydata&bitmask;
		// Get population of ones and zeroes
		unsigned int ones = __ballot(mybit);
		unsigned int zeroes = ~ones;

		// Switch ping-pong buffers
		offset ^= SIZE;

		// Do zeroes, then ones
		if (!mybit)
		{
			mypos = __popc(zeroes&thrmask);
		}
		else {      // Threads with a one bit
					// Get my position in ping-pong buffer
			mypos = __popc(zeroes) + __popc(ones&thrmask);
		}

		// Move to buffer  (or use shfl for cc 3.0)
		shared_data[mypos - 1 + offset] = mydata;
		// Repeat for next bit
		bitmask <<= 1;

		device_data[tib] = shared_data[tib + offset];
	}
		

}

void WriteFile(vector<double> vector) {
	ofstream myfile;
	myfile.open("out.txt");
	for each (double var in vector)
	{
		myfile << var << " ";
	}
	myfile.close();
}
vector<int> ReadFile(string filepath) {
	vector<int> vector;
	ifstream inputFile(filepath);
	double value;

	// read the elements in the file into a vector  
	while (inputFile >> value) {
		vector.push_back(value * 100);
	}
	for each (int var in vector)
	{
		cout << var << " ";
	}
	inputFile.close();
	return vector;
}

int main(int argc, char * argv[])
{
	cout << argv[1] << endl;

	std::ifstream fin(argv[1]);
	
	std::vector<int> host_double(9);
	std::vector<int> host_double2(9);

	float totalTime = 0;
	float milliseconds = 0;

	//Time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (!fin) {
		cerr << "Datoteka ne obstaja " << argv[1] << endl;
	}
	else {
		cout << "Datoteka obstaja " << argv[1] << endl;
		host_double = ReadFile(argv[1]);

		//WriteFile(host_double);
	}
	/*int size = host_double.size();*/
	
	//int* device = NULL;
	//int* device2 = NULL;
	//
	//int size = host_double.size();

	//hipMalloc((void**)&device, size * sizeof(int));
	//hipMalloc((void**)&device2, size * sizeof(int));

	////num_list 32 

	//hipMemcpy(device_data, host_double.data(), size * sizeof(int), hipMemcpyHostToDevice);
	//
	//radix_sort << <(size + size - 1) / size, size >> >();
	////ParalelRadixSort << <(size + size - 1) / size, size >> >();
	////gpucopy << <(size + size - 1) / size, size >> >(device, device2);
	//
	//hipDeviceSynchronize();

	//hipMemcpy(host_double.data(), device_data, size * sizeof(int), hipMemcpyDeviceToHost);

	//

	//cout << endl << "host data" << endl;
	//for each (int var in host_double)
	//{
	//	double d = (double)var / 10;
	//	cout << d << " " << endl;
	//}

	//hipFree(device);

	std::vector<int> v(10);

	unsigned int hdata[SIZE];


		int size = host_double.size();
		//vector.resize(size);

		std::copy(host_double.begin(), host_double.end(), hdata);
		//int *a = vector.data();

		// Copy data from host to device
		hipMemcpyToSymbol(HIP_SYMBOL(device_data), hdata, SIZE * sizeof(unsigned int));

		// Execution time measurement, that point starts the clock
		hipEventRecord(start);
		ParalelRadixSort << <1, SIZE>> >();

		// Execution time measurement, that point stops the clock
		hipEventRecord(stop);

		// Make kernel function synchronous
		hipDeviceSynchronize();

		hipEventElapsedTime(&milliseconds, start, stop);
		totalTime += milliseconds;
		//milliseconds = 0;

		// Copy data from device to host
		hipMemcpyFromSymbol(hdata, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));

		/*std::vector<int> v(hdata, hdata + sizeof hdata / sizeof hdata[0]);
		v.resize(size);*/
		milliseconds = 0;
	

	/*printf("TIME %4.2fs", milliseconds);
	printf("\n");
	printf("Effective Bandwidth (GB/s): %fn", SIZE * 4 * 3 / milliseconds / 1e6);
	printf("\n");*/
	cout << endl;
	cout << "Host Data" << endl;
	/*for each (int var in v)
	{
		double d = (double)var / 100;
		cout << d << " " << endl;
	}*/
	for (int i = 0; i < SIZE; i++) {
		double d = (double)hdata[i] / 100;
		cout << d << endl;
	}
	
	cout << endl;
	cout << "TIME %4.2fs " << totalTime << endl;

    return 0;
}







//
//#define nTPB 512
//
//__global__ void gpucopy(int* src, int* dst)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;;
//	dst[i] = src[i];
//}
//
//int main()
//{
//	const int arraySize = 500; // >= 1025 will fail on my system!
//
//	int* data1 = new int[arraySize];
//	int* data2 = new int[arraySize];
//	// Initialized both data1 and data2
//	// ... 
//	for (int i = 0; i < arraySize; i++) {
//		data1[i] = 2 * i;
//		//cout << data1[i] + " ";
//	}
//		
//
//	int* dev_data1 = NULL;
//	int* dev_data2 = NULL;
//	// Initialized both dev_data1 and dev_data2
//	// ... 
//	hipMalloc(&dev_data1, arraySize * sizeof(int));
//	hipMalloc(&dev_data2, arraySize * sizeof(int));
//
//	// copy data1 to device
//	hipMemcpy(dev_data1, data1, arraySize * sizeof(int), hipMemcpyHostToDevice);
//
//	//// copy dev_data1 to dev_data2 with gpu
//	//gpucopy << <1, arraySize >> >(dev_data1, dev_data2);
//	gpucopy << <(arraySize + nTPB - 1) / nTPB, nTPB >> >(dev_data1, dev_data2);
//
//	// copy dev_data2 to data
//	hipMemcpy(data2, dev_data2, arraySize * sizeof(int), hipMemcpyDeviceToHost);
//
//
//	for (int i = 0; i<arraySize; i++)
//		if (data2[i] != data1[i])
//			cout << "Error: data is different - data2[" << i << "] is " << data2[i] << endl;
//
//	return 0;
//}
//
//
//
