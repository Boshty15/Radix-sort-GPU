#include "hip/hip_runtime.h"



//#pragma once
//#ifdef __INTELLISENSE__
//void __syncthreads();
//#endif


#include "hip/hip_runtime.h"
#include ""

//#ifndef __CUDACC_RTC__ 
//#define __CUDACC_RTC__
//#endif
//#pragma once
//#ifdef __INTELLISENSE__
//void __syncthreads();
//
//#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "kernel.h"
#include <iostream>     
#include <fstream>
#include <vector>
#include <bitset>
#include <time.h>
#include <chrono>
#include "hip/device_functions.h"
#include <math.h> 


//#define SIZE 32
#define BIT 32
//
using namespace std;
using namespace chrono;
//
//__device__ unsigned int device_data[SIZE];
//__device__ int ddata_s[SIZE];
//
////Test v gpu in nazaj
//__global__ void gpucopy(int* src, int* dst)
//{
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	int stride = blockDim.x * gridDim.x;
//	for (int i = index; i < SIZE; i += stride) {
//		if(src[i] > 50)
//			src[i] = dst[i] + 100;
//		else 
//			src[i] = dst[i] - 100;
//	}
//		
//}
//

//
//
//void WriteFile(vector<double> vector) {
//	ofstream myfile;
//	myfile.open("out.txt");
//	for each (double var in vector)
//	{
//		myfile << var << " ";
//	}
//	myfile.close();
//}

//
//int main(int argc, char * argv[])
//{
//	cout << argv[1] << endl;
//
//	std::ifstream fin(argv[1]);
//	
//	std::vector<unsigned int> host_double(7);
//	std::vector<int> host_double2(7);
//
//	float totalTime = 0;
//	float milliseconds = 0;
//
//	//Time
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//
//	if (!fin) {
//		cerr << "Datoteka ne obstaja " << argv[1] << endl;
//	}
//	else {
//		cout << "Datoteka obstaja " << argv[1] << endl;
//		host_double = ReadFile(argv[1]);
//
//		//WriteFile(host_double);
//	}
//	int size = host_double.size();
//	
//	int* device = NULL;
//	int* tmp_0 = NULL;
//	int* tmp_1 = NULL;
//	
//	hipMalloc((void**)&device, size * sizeof(int));
//	hipMalloc((void**)&tmp_0, size * sizeof(int));
//	hipMalloc((void**)&tmp_1, size * sizeof(int));
//
//	int blockSize = 4;
//	int numBlocks = (SIZE + blockSize - 1) / blockSize;
//	
//	//string binary = bitset<16>(222).to_string(); //to binary
//	//cout << binary << "\n";	
//
//	//unsigned long decimal = bitset<16>(binary).to_ulong();
//	//cout << decimal << "\n";
//
//	//num_list 32 
//
//	hipMemcpy(device,&*host_double.data(), size * sizeof(int), hipMemcpyHostToDevice);
//	
//	//radixSort << <numBlocks, blockSize>> >(device, SIZE, size, tmp_1);
//	//ParalelRadixSort << <(size + size - 1) / size, size >> >();
//	gpucopy << <numBlocks, blockSize>> >(device, device);
//	
//	hipDeviceSynchronize();
//
//	hipMemcpy(&*host_double.data(), device, size * sizeof(int), hipMemcpyDeviceToHost);
//
//	cout << endl << "host data" << endl;
//	for each (unsigned int var in host_double)
//	{
//		double d = (double)var / 100;
//		cout << d << " " << endl;
//	}
//
//	hipFree(device);
//	hipFree(tmp_0);
//	hipFree(tmp_1);
//
//	//std::vector<int> v(10);
//
//	//unsigned int hdata[SIZE];
//
//
//	//	int size = host_double.size();
//	//	//vector.resize(size);
//
//	//	std::copy(host_double.begin(), host_double.end(), hdata);
//	//	//int *a = vector.data();
//
//	//	// Copy data from host to device
//	//	hipMemcpyToSymbol(HIP_SYMBOL(device_data), hdata, SIZE * sizeof(unsigned int));
//
//	//	// Execution time measurement, that point starts the clock
//	//	hipEventRecord(start);
//	//	ParalelRadixSort << <1, SIZE>> >();
//
//	//	// Execution time measurement, that point stops the clock
//	//	hipEventRecord(stop);
//
//	//	// Make kernel function synchronous
//	//	hipDeviceSynchronize();
//
//	//	hipEventElapsedTime(&milliseconds, start, stop);
//	//	totalTime += milliseconds;
//	//	//milliseconds = 0;
//
//	//	// Copy data from device to host
//	//	hipMemcpyFromSymbol(hdata, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));
//
//	//	/*std::vector<int> v(hdata, hdata + sizeof hdata / sizeof hdata[0]);
//	//	v.resize(size);*/
//	//	milliseconds = 0;
//	//
//
//	///*printf("TIME %4.2fs", milliseconds);
//	//printf("\n");
//	//printf("Effective Bandwidth (GB/s): %fn", SIZE * 4 * 3 / milliseconds / 1e6);
//	//printf("\n");*/
//	//cout << endl;
//	//cout << "Host Data" << endl;
//	///*for each (int var in v)
//	//{
//	//	double d = (double)var / 100;
//	//	cout << d << " " << endl;
//	//}*/
//	//for (int i = 0; i < SIZE; i++) {
//	//	double d = (double)hdata[i] / 100;
//	//	cout << d << endl;
//	//}
//	//
//	//cout << endl;
//	//cout << "TIME %4.2fs " << totalTime << endl;
//
//    return 0;
//}



#define SIZE (33 * 1024)
#define LOOPS 1
#define UPPER_BIT 31
#define LOWER_BIT 0
int getMax(unsigned int* arr, int n)
{
	int mx = arr[0];
	for (int i = 1; i < n; i++)
		if (arr[i] > mx)
			mx = arr[i];
	return mx;
}

// A function to do counting sort of arr[] according to
// the digit represented by exp.
void countSort(unsigned int* arr, int n, int exp)
{
// output array
	vector<unsigned int> ve(n);
	unsigned int* output = ve.data();
	int i, count[10] = { 0 };

	// Store count of occurrences in count[]
	for (i = 0; i < n; i++)
		count[(arr[i] / exp) % 10]++;

	// Change count[i] so that count[i] now contains actual
	//  position of this digit in output[]
	for (i = 1; i < 10; i++)
		count[i] += count[i - 1];

	// Build the output array
	for (i = n - 1; i >= 0; i--)
	{
		output[count[(arr[i] / exp) % 10] - 1] = arr[i];
		count[(arr[i] / exp) % 10]--;
	}

	// Copy the output array to arr[], so that arr[] now
	// contains sorted numbers according to current digit
	for (i = 0; i < n; i++)
		arr[i] = output[i];
}

// The main function to that sorts arr[] of size n using 
// Radix Sort
void serialRadixSort(unsigned int* arr, int n)
{
	// Find the maximum number to know number of digits
	int m = getMax(arr, n);

	// Do counting sort for every digit. Note that instead
	// of passing digit number, exp is passed. exp is 10^i
	// where i is current digit number
	for (int exp = 1; m / exp > 0; exp *= 10)
		countSort(arr, n, exp);
}

// A utility function to print an array


__device__ unsigned int device_data[SIZE];

// naive warp-level bitwise radix sort
__global__ void radixSortParralel(unsigned int* sort_tmp, unsigned int* sort_tmp_1) {
	const int num_lists = SIZE;
	const int num_element = 10;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;   
	//printf("\n%d num list: %d" ,tid, num_lists );
	//printf("\n num element: %d" , num_element);
	if (sort_tmp == NULL) {
		printf("\nsort_tmp is null");
	}
	

    for (int bit = 0; bit < BIT; bit++) {
		const int bit_mask = (1 << bit);
		int base_cnt_0 = 0, base_cnt_1 = 0;
		//printf("\ntest1 + %d",tid );
		for (int i = 0; i < num_element; i += num_lists) {
			int elem = sort_tmp[i + tid];
			
			//printf("\ntest %d + %d bit %d",tid, elem, bit_mask);
			if ((elem & bit_mask) > 0) {
				sort_tmp_1[base_cnt_1 + tid] = elem;
				base_cnt_1 += num_lists;
				//printf("\n element 1 %d = %d",i, elem);
			}
			else {
				sort_tmp[base_cnt_0 + tid] = elem;
				base_cnt_0 += num_lists;
				//printf("\n element 0 %d = %d",i, elem);
			}
		}
		//copy data back to source - first the zero list
		for (int i = 0; i < base_cnt_1; i += num_lists) {
			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
			//printf("\nsort tmp: %d", sort_tmp_0[i + tid]);
		}
	}
	//hipDeviceSynchronize();
	//__syncthreads();

	//for (int i = 0; i < num_element; i++) {
	//	printf("\n%d", sort_tmp[i]);
	//}
}

__global__ void radixSort(unsigned int* data, unsigned int* data_tmp) {
	//__shared__ volatile unsigned int shared_data[SIZE * 2];
	// load from global into shared variable
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < SIZE) {
		//data[tid] = data[tid] + 1000;



		tid += blockDim.x * gridDim.x;
	}

}
void print(unsigned int* arr, int n)
{
	for (int i = 0; i < n; i++)
		cout << (double)arr[i] / 100 << endl;
}

void menu() {
	cout << "Izberi: " << endl;
	cout << "0 Exit " << endl;
	cout << " 1 Serial sort" << endl;
	cout << " 2 Parrallel sort" << endl;
}

vector<unsigned int> ReadFile(string filepath) {
	vector<unsigned int> vector;
	ifstream inputFile(filepath);
	double value;

	//read the elements in the file into a vector  
	while (inputFile >> value) {
		//bitset<32>tmp(value *100);
		//cout << tmp << " ";
		vector.push_back(value * 100);
		//cout << (int)(tmp.to_ulong()) << " "; // konvert nazaj to int
	}
	/*for each (unsigned int var in vector)
	{
	cout << var << " " << endl;
	}*/
	inputFile.close();
	return vector;
}

int main(int argc, char * argv[]) {

	int blockSize = 32;
	int numBlocks = (SIZE + blockSize - 1) / blockSize;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float totalTime = 0;

	std::ifstream fin(argv[1]);
	std::vector<unsigned int> host_double(SIZE);

	unsigned int host_data[SIZE];

	if (!fin) {
		cerr << "Datoteka ne obstaja " << argv[1] << endl;
	}
	else {
		cout << "Datoteka obstaja " << argv[1] << endl;
		host_double = ReadFile(argv[1]);
		
		//WriteFile(host_double);
	}
	int size = host_double.size();

	int izbiraAlg;
	menu();
	cin >> izbiraAlg;
	if (izbiraAlg == 0) {

	}else if (izbiraAlg == 1) {

		//Serial radix sort

		unsigned int* da = host_double.data();
		//int arr[] = { 170, 45, 75, 90, 802, 24, 2, 66, 170, 45, 75, 90, 802, 24, 2, 66 };
		int n = sizeof(host_double) / sizeof(host_double[0]);
		//auto t1 = high_resolution_clock::now();
		high_resolution_clock::time_point t1 = high_resolution_clock::now();
		serialRadixSort(da, size);
		high_resolution_clock::time_point t2 = high_resolution_clock::now();
		auto duration = duration_cast <milliseconds> (t2 - t1).count();
		//auto t2 = high_resolution_clock::now();
		//auto diff = duration_cast<duration<double>>(t2 - t1);
		// now elapsed time, in seconds, as a double can be found in diff.count()
		//long ms = (long)(1000 * diff.count());
		cout << endl << "Serial radix sort " << endl;
		//print(da, size);
		for (int i = 0; i < size - 1; i++) {
			if (host_double[i] > host_double[i + 1]) {
				printf("sort error at, hdata[%d] = %d, hdata[%d] = %d\n", i, host_double[i], i + 1, host_double[i + 1]);
				return 1;
			}
		}
		cout << "Total time: " << duration << "ms" << endl;
		cout << "Success" << endl;

	}
	else {

		//Parrallel radix sort
		cout << endl;
		//unsigned int t = decToBinary(10);
		//unsigned int tmp22;
		/*bitset<32>tmp(65.55);
		cout << tmp << endl;*/
			
		cout << "Parrallel sort " << endl;
		std::copy(host_double.begin(), host_double.end(), host_data);
		cout << "Not sorted!" << endl;
		print(host_data, size);

		unsigned int* ddata;
		unsigned int* ddata_tmp;
		//

		//for (int lcount = 0; lcount < LOOPS; lcount++) {
		//	unsigned int range = 1U << UPPER_BIT;
		//	//for (int i = 0; i < SIZE; i++) host_data[i] = rand() % range;
			//hipMemcpyToSymbol(HIP_SYMBOL(device_data), host_data, SIZE * sizeof(unsigned int));
			hipMalloc((void**)&ddata_tmp, SIZE * sizeof(unsigned int));
			hipMalloc((void**)&ddata, SIZE * sizeof(unsigned int));
			hipMemcpy(ddata, host_data, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);			

			high_resolution_clock::time_point start = high_resolution_clock::now();
			radixSortParralel << <1, 10 >> >(ddata, ddata_tmp);
			//radixSort << <(numBlocks, blockSize) >> > (ddata, ddata_tmp);
			high_resolution_clock::time_point stop = high_resolution_clock::now();

			hipMemcpy(host_data, ddata, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);

			//hipMemcpyFromSymbol(host_data, HIP_SYMBOL(device_data), SIZE * sizeof(unsigned int));
			auto durationParralel = duration_cast <milliseconds> (start - stop).count();
		//	/*for (int i = 0; i < SIZE - 1; i++) {
		//	if (host_data[i] > host_data[i + 1]) {
		//	printf("sort error at loop %d, hdata[%d] = %d, hdata[%d] = %d\n", lcount, i, host_data[i], i + 1, host_data[i + 1]);
		//	return 1;
		//	}
		//	}*/
		//	cout << " Sorted data: " << endl;
		//	for (int i = 0; i < size; i++) {
		//		cout << i << "     " <<(double)host_data[i] / 100 << endl;
		//	}
		//	cout << " Time parralel sort: " << durationParralel << endl;
		//}
			cout << endl << "Sorted" << endl;
			print(host_data, size);
		
	}
	
	
	return 0;
}
